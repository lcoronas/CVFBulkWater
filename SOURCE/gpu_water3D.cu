#include "hip/hip_runtime.h"

/****************************************************************/
/*								*/
/*			Bulk FS Water				*/
/*								*/
/*								*/
/****************************************************************/

#include <stdlib.h>
#include <stdio.h>
#include <stdint.h>
#include <stdbool.h>
#include <math.h>
#include <time.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <sys/time.h>
#include <hip/hip_vector_types.h>
#include <string.h>


#include "mersenne.h"
#include "chessboard.h"

#define delta(X, Y)  ((X) == (Y)) // ? 1 : 0)
#define theta(X, Y)  ((X) <= (Y)) // ? 1 : 0)

//#define CPU
#define GPU

/* Lattice NX*NY*NZ, adjust */
#define NX 32            
#define NY 32
#define NZ 32

/* Size of the random number arrays (GPU calculation) */
#define N_RANDOM 500 

/* How many iterations for each measurements */
int N_MEASURE;

int NLOOPS;

int NEQUILIBRIUM;

/* 1 MC step = NMETROPOLIS + NCLUSTER */
int NMETROPOLIS, NCLUSTER;

void MonteCarlo_Step(int time, int &loop, int &offset, int &loop_chess, int &offset_chess, int nblocks, int nthreads, int *order,
     FILE* logfile, int *cluster_size, FILE* logsize);


/* Model parameters */
#define J 		1.f
#define J_hb 		0.5f
#define J_sig 	0.08f
#define v_hb 		0.6f

#define q	6
#define NHBMAX 	4
#define ARMS		6

/* Number of parallel threads per block */
//#define NTHREADS 1024
#define NTHREADS 512
//#define NTHREADS 64
//#define NTHREADS 36
//#define NTHREADS 1

int flag_config;
int flag_distances;

float T, P;  //Temperature, Pressure
double V;    //Volume

//   xorshift1024*Phi pseudorandom number generator
uint64_t xor_s[16]; 
int xor_p;
void initRand(uint64_t  seed);
uint64_t nextRand(void);
double uniformDoubleRand(void);

/// This macro activates the check for errors with the GPU (may influence performance)
#define CUDA_ERROR_CHECK
#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )

typedef uint8_t byte;

int deviceID;
void __cudaSafeCall( hipError_t err, const char *file, const int line );
void __cudaCheckError( const char *file, const int line );

__global__ void gpu_RNG_setup (hiprandState * state, uint * seed);
__global__ void gpu_RNG_generate ( hiprandState* globalState, float * Rnd, int n_rand);
__global__ void gpu_update (byte * dev_s, byte * dev_active_bond, byte * dev_nhb,
   float * dev_Rnd, float * dev_Rnd_spin, int indices, float P, float T, float V, int offset, byte random_index, int flag_chessboard);
   
void cpu_update();
   
//#define cpu_METROPOLIS
#define gpu_METROPOLIS

__global__ static void _rand(float* vec, uint* z1, uint* z2, uint* z3, uint* z4);
__device__ static float HybridTaus(uint& z1, uint& z2, uint& z3, uint& z4);

void compute_indices(int *a,int *b);

hiprandState* devStates;

/* van der Waals interacion */
float * distance;
int * frequency;
float distance_PBC(int x1, int y1, int z1, int x2, int y2, int z2);
int num_distances;
float R_cutoff, LJ_InfiniteBarrier=0;

void volume_step(double * r_cell, double * energy, double * volume);
void calculate_energy_vdW (double * energy);

/* sigma variables storage */
byte * s, * dev_s, * nhb, * dev_nhb;
double beta;

/* eta variables storage */
byte * active_bond, * dev_active_bond;
int * rand_Chessboard;

#define ACTIVE 1
#define NON_ACTIVE 0

/* Neighbour index arrays, to be filled at the beginning  */
int * xup, * yup, * zup, 
    * xdn, * ydn, * zdn, 
    * dev_xup, * dev_yup, * dev_zup,
    * dev_xdn, * dev_ydn, * dev_zdn;
float * dev_Rnd, * Rnd;
float * dev_Rnd_spin, * Rnd_spin;
float * dev_Rnd_array, * Rnd_array;
float * acc, * dev_acc;

void update(void);
void measure(int iter, FILE *f);

uint * seed_array, * dev_seed_array;

double r_cell = 1.0001;
double energy_vdW = 0;  //van der Waals energy

double E = 0, E2 = 0, M = 0, M2 = 0;

// save configurations
//int flag_save_config;
//int * logaritmic_times;
//int max_power, num_of_times;
//bool is_logaritmic_time(int);

/* autocorrelation function calculation */

int flag_correlation;

#define numcorrelators 32
#define p 16
#define m_ 2

// correlator
struct correlator {
	double ** shift;
	double ** correlation;
	long int ** ncorrelation;
	double * accumulator;
	int * naccumulator;
	int * insertindex;
	double * t;
	double * f;
	int npcorr;
	int kmax;
	double accval;
};

void initialize(int N, struct correlator mol[]);
void add(struct correlator mol[], int i, double w, int k);
void evaluate(struct correlator mol[], int N, int norm);
void printCorr(struct correlator corr[], char filename[],int total_correlators);
//void HistogramTest(struct correlator mol[], int N, int norm);

struct correlator *Corr;
struct correlator *corr;

int *histo_sigma;

/* Wolff CPU cluster algorithm */
//#define WOLFF
double Jeff;
void cluster_step();
int site (int arm, int cell);
void shuffle ();
int cluster_poke ();
int add_to_cluster (int arm, int cell);
void update_cluster (int arm, int cell);
int * order, * neigh_arm, * neighbor;
bool * is_cluster;
int cluster_size;
float pJ_hb, pJ_s;

/* Swendsen-Wang GPU cluster algorithm */
#define SWENDSEN_WANG
#define SW_links_per_cell 18  
   // xup, yup, zup + 15 sigma bonds = 18
   // in each call to gpu cluster kernels, a cell works on 18 links between spins
float * dev_Rnd_cluster, * dev_Rnd_update;
byte * dev_delta, * cpu_delta, * dev_converges, *converges;
int * dev_label, * cpu_label, *dev_prev_label;

__global__ void gpu_initialize_cluster_variables (byte *dev_delta, int *dev_label, int *dev_prev_label);
__global__ void gpu_create_cluster( float* dev_Rnd_cluster, byte* dev_delta, byte* dev_s, 
                                    byte * dev_active_bond, float pJ_hb, float pJ_s, float Jeff);
__global__ void gpu_cluster_scanning_covalent (byte * dev_delta, int * dev_label);
__global__ void gpu_cluster_scanning_sigma (byte * dev_delta, int * dev_label);
__global__ void gpu_cluster_analysis(int *dev_label, int spin);
__global__ void gpu_update_cluster(int * dev_label, byte * dev_s, int offset_update, float * dev_Rnd_update);
__global__ void gpu_convergence_test(int * dev_label, int * dev_prev_label, byte * converges);

int offset_update, loop_update;
bool cluster_converges;
int max_SW_scans=-1;
int max_cluster_size, max_cluster_size2, tmp_max_cluster_size, num_clusters;
int *SW_cluster_size;
float average_cluster_size;

/* Swendsen-Wang CPU algorithm */
//#define cpu_SWENDSEN_WANG
int ** SW_bonded;
int * cpu_SW_L, * cpu_SW_N, * neighbor_spin, * SW_new_spin, * HK_label;// * cpu_SW_L2, * cpu_SW_L3;

void cpu_SW_initialize();
void cpu_SW_neighboring_spins(int i);
void cpu_SW_update(int* label);

//void dfs_rec ( int u, int this_label );
//int *visited;

/* chessboard / checkerboard algorithm: update of eta variables */
int flag_chessboard;

int * devChessboard_state, * devChessboard_edge, * devChessboard_vertex, * devSolutions;
__global__ void gpu_set_cubes(int * devChessboard_edge, int * devChessboard_vertex);

__global__ void gpu_Chessboard_start_cubes(int N, float * dev_Rnd_cubeFlip,
 int * devSolutions, byte * dev_active_bond, int * devChessboard_edge,
 int * devChessboard_vertex, int * devChessboard_state );
 
__global__ void gpu_Chessboard_set_state( int N, byte * dev_active_bond,
 int * devChessboard_vertex, int * devChessboard_state );

__global__ void gpu_Chessboard_set_active_bonds (byte * dev_s, int N, double Jeff, double beta,
     float * rand0, float * rand1, int offset_chess,
     int * devSolutions, byte * dev_active_bond, int * devChessboard_edge,
     int * devChessboard_vertex, int * devChessboard_state);

float * dev_Rnd_chess, * dev_Rnd_cubeFlip;
int offset_chess, loop_chess;

//cluster statisitcs

void print_cluster_statistics(int time, FILE * logsize, int* label);
int * cluster_size_;
byte * Xcoord, *Ycoord, *Zcoord; 
int is_percolating_cluster(int *label, int i);


//////////////////////
///////  MAIN
//////////////////////

int main ( int argc, char * argv[] ) 
{
  int i,j,n_loops=0,loop=0,offset=0,n_equilibrium=0;
  uint64_t seed;
  char input_data[200];
  char output_data[200];
  char input_config[200];
  char output_config[200];
  char log_file[200];
  FILE *idata, *odata, *iconfig, *oconfig, *logfile;

  int total_global_memory = 0;

  /* Read the input */
     sprintf(input_data,  "input_data");
     idata = fopen(input_data, "r");
     if(idata == NULL){
       fprintf(stderr,"Can't open input file\n");
       exit(1);
     }

     char parameter[50];
     
     if (fscanf(idata,"%s %f", &parameter [0], &P) != 2){
          fprintf(stderr,"Error [Pressure]: Can't read input file\n");
          fclose(idata);
          exit(1);
     }

     if (fscanf(idata,"%s %f", &parameter [0], &T) != 2){
          fprintf(stderr,"Error [Temperature]: Can't read input file\n");
          fclose(idata);
          exit(1);
     }

     if (fscanf(idata,"%s %ld", &parameter [0], &seed) != 2){
          fprintf(stderr,"Error[Seed]: Can't read input file\n");
          fclose(idata);
          exit(1);
     }

     if (fscanf(idata,"%s %d", &parameter [0], &NMETROPOLIS) != 2){
          fprintf(stderr,"Error[Metropolis_Steps]: Can't read input file\n");
          fclose(idata);
          exit(1);
     }

     if (fscanf(idata,"%s %d", &parameter [0], &NCLUSTER) != 2){
          fprintf(stderr,"Error[Cluster_Steps]: Can't read input file\n");
          fclose(idata);
          exit(1);
     }

     if (fscanf(idata,"%s %d", &parameter [0], &NEQUILIBRIUM) != 2){
          fprintf(stderr,"Error[Equilibration_Steps]: Can't read input file\n");
          fclose(idata);
          exit(1);
     }

     if (fscanf(idata,"%s %d", &parameter [0], &NLOOPS) != 2){
          fprintf(stderr,"Error[Sampling_Steps]: Can't read input file\n");
          fclose(idata);
          exit(1);
     }

     if (fscanf(idata,"%s %d", &parameter [0], &N_MEASURE) != 2){
          fprintf(stderr,"Error[Sampling_Interval]: Can't read input file\n");
          fclose(idata);
          exit(1);
     }

     if (fscanf(idata,"%s %f", &parameter [0], &R_cutoff) != 2){
          fprintf(stderr,"Error[R_cutoff]: Can't read input file\n");
          fclose(idata);
          exit(1);
     }

     if (fscanf(idata,"%s %d", &parameter [0], &flag_config) != 2){
          fprintf(stderr,"Error[Flag_config]: Can't read input file\n");
          fclose(idata);
          exit(1);
     }

     if (fscanf(idata,"%s %d", &parameter [0], &flag_distances) != 2){
          fprintf(stderr,"Error[Flag_distances]: Can't read input file\n");
          fclose(idata);
          exit(1);
     }

     if (fscanf(idata,"%s %d", &parameter [0], &flag_correlation) != 2){
          fprintf(stderr,"Error[Flag_correlation]: Can't read input file\n");
          fclose(idata);
          exit(1);
     }

     if (fscanf(idata,"%s %d", &parameter [0], &flag_chessboard) != 2){
          fprintf(stderr,"Error[Flag_chessboard]: Can't read input file\n");
          fclose(idata);
          exit(1);
     }

     if (fscanf(idata,"%s %d", &parameter [0], &deviceID) != 2){
          fprintf(stderr,"Error[Device_ID]: Can't read input file\n");
          fclose(idata);
          exit(1);
     }

     fclose(idata);

  beta = 1./T;
  n_loops = NLOOPS;
  n_equilibrium = NEQUILIBRIUM;
  Jeff = J_hb - P*v_hb;

#ifdef CPU
  sprintf(output_data,  "test_cpu.out");
#endif
#ifdef GPU
  sprintf(output_data,  "data.out");
  sprintf(output_config, "config.out");
  sprintf(input_config, "input_config");
#endif

/* Verify that only one, cpu or gpu Metropolis, is enabled */

#ifdef gpu_METROPOLIS
  #ifdef cpu_METROPOLIS
       fprintf(stderr,"ERROR!! Both Metropolis CPU and GPU algorithms are active!!\n");
       fprintf(stderr,"Please check source file\n");
       exit(1);
  #endif
#endif

/* Verify that only one, Wolff, cpu SW, or gpu SW, is enabled */
#ifdef WOLFF
   #ifdef SWENDSEN_WANG
       fprintf(stderr,"ERROR!! Both Swendsen-Wang and Wolff cluster algorithms are active!!\n");
       fprintf(stderr,"Please check source file\n");
       exit(1);
   #endif
   #ifdef cpu_SWENDSEN_WANG
       fprintf(stderr,"ERROR!! Both Swendsen-Wang and Wolff cluster algorithms are active!!\n");
       fprintf(stderr,"Please check source file\n");
       exit(1);
   #endif
#endif

#ifdef cpu_SWENDSEN_WANG
   #ifdef SWENDSEN_WANG
       fprintf(stderr,"ERROR!! Both Swendsen-Wang CPU and GPU algorithms are active!!\n");
       fprintf(stderr,"Please check source file\n");
       exit(1);
   #endif
#endif

  odata = fopen(output_data,"w");       /* open output file */
  if(odata == NULL){
     fprintf(stderr,"Can't create output file\n");
     exit(1);
  }

  sprintf(log_file,  "water3D.log");     /* open log file */
  logfile = fopen(log_file, "w");  
  if(logfile == NULL){
     fprintf(stderr,"Can't create log file\n");
     exit(1);
  }

  fprintf(logfile, "++++ Welcome to water3D log file ++++\n");

  seed_mersenne( seed );    //initialize mersenne
  initRand( seed );         //initialize xorsifht1024*Phi
  
  if ((NX*NY*NZ) %  NTHREADS != 0){
    fprintf(stderr,"Error, NX*NY*NZ %d is not a multiple of NTHREADS %d \n",NX*NY*NZ, NTHREADS);
    return 1;
  }

#ifdef GPU

    int total_devices;  /* set GPU device */
    CudaSafeCall(hipGetDeviceCount(&total_devices));
    if ( deviceID < 0 || deviceID >=total_devices ){
       fprintf(stderr,"Error: Tried to execute in GPU deive ID = %d\n",deviceID);
       fprintf(stderr,"\tTotal number of devices is %d\n",total_devices);
       return 1;
    }
    CudaSafeCall(hipSetDevice(deviceID));

#endif
    int maxMB = 5000;     // Avoid excess of global memory storage (device). The limit is hardware dependent and may be modified.

  int total_RNG_memory = (5*N_RANDOM+SW_links_per_cell)*NX*NY*NZ;  
           //4 vectors of sizes N_RANDOM (Metropolis & Chessboard) + 1 vector of size 18 (Swendsen-Wang)
  
  fprintf(logfile,"RNG allocated memory: %d MB\n", (total_RNG_memory*sizeof(float))/(1024*1024));
  
  if((total_RNG_memory*sizeof(float))/(1024*1024) > maxMB) {
    fprintf(stderr,"Error, memory allocated > %d MB.\n",maxMB);
    return 1;
  }
  
  // NX*NY*NZ of the lattice and initial volume
  V = NX*NY*NZ*r_cell*r_cell*r_cell;

  fprintf(logfile," ++++++++++++++++++++++++++++++++++++++++++\n");
  fprintf(logfile,"Source file: %s\n",__FILE__);
  fprintf(logfile," 3D water model, %d x %d x %d lattice, N = %d, P = %1.2f, T = %1.2f\n",NX,NY,NZ,NX*NY*NZ,P,T);
  fprintf(logfile," J_hb = %1.2f, J_sigma = %1.2f, v_HB = %1.2f\n",J_hb,J_sig,v_hb);
  fprintf(logfile," Flag configuration = %d\n",flag_config);
  fprintf(logfile," Monte Carlo Step: %d Metropolis + %d Cluster\n",NMETROPOLIS,NCLUSTER);
#ifdef gpu_METROPOLIS
  fprintf(logfile," Metropolis: Parallel GPU algorithm\n");
#endif
#ifdef cpu_METROPOLIS
  fprintf(logfile," Metropolis: Sequential CPU algorithm\n");
#endif
#ifdef WOLFF
  fprintf(logfile," Cluster: Wolff Recursive CPU algorithm\n");
#endif
#ifdef SWENDSEN_WANG
  fprintf(logfile," Cluster: Swendsen-Wang Parallel GPU algorithm\n");
#endif
#ifdef cpu_SWENDSEN_WANG
  fprintf(logfile," Cluster: Swendsen-Wang Sequential CPU algorithm\n");
#endif
  fprintf(logfile," Device = %d\n",deviceID);
  fprintf(logfile," %d equilibration, %d updates/measurement, %d updates\n",
	 n_equilibrium, N_MEASURE, n_loops);
  fprintf(logfile," Output file %s\n",output_data);
  fprintf(logfile," Random seed %ld\n", seed );

  int dist_index=0;      //index to build distance and frequency
  num_distances= (int) 50;    //guess value for the total number of distances
  distance = (float*)calloc(num_distances,sizeof(float));
  frequency = (int*)calloc(num_distances,sizeof(int));

#ifdef GPU
  int nthreads = NTHREADS;
  int nblocks = NX*NY*NZ/nthreads; /// Synchronous blocks

  if (nthreads > 1024){   /* Hardware dependent. May be modified */
     fprintf(stderr,"Excessive number of threads. Reduce NTHREADS.\n");
     exit(-1);
  }
  if (nblocks > 65000){   //* Hardware dependent. May be modified */
     fprintf(stderr,"Excessive number of blocks. Increase the number of threads.\n");
     exit(-1);
  }
#endif  

  s = (byte*)calloc(ARMS*NX*NY*NZ,sizeof(byte));	// sigma variables
  
  nhb = (byte*)calloc(NX*NY*NZ,sizeof(byte));		// controls nhb per cell (if chessboard OFF)
 
  seed_array = (uint*)calloc((NX*NY*NZ),sizeof(uint));  // array of seed for random number generator

  active_bond = (byte*)calloc(ARMS*NX*NY*NZ,sizeof(byte));  // eta variables

  srand(seed);  
  
  for (i=0; i<(NX*NY*NZ); ++i) {
    if(i == 0)
      seed_array[i] = rand();
    else
      seed_array[i] = rand() ^ seed_array[i-1];
  }

#ifdef GPU  

  /* Allocate arrays in the GPU */

  total_global_memory += (NX*NY*NZ)*sizeof(hiprandState);
  CudaSafeCall(hipMalloc( (void**)&devStates, (NX*NY*NZ)*sizeof(hiprandState)));

  total_global_memory += (NX*NY*NZ)*sizeof(uint);
  CudaSafeCall(hipMalloc( (void**)&dev_seed_array, (NX*NY*NZ)*sizeof(uint)));

          hipError_t err = hipGetLastError();
          if (err != hipSuccess)
          {
             fprintf(stderr, "Allocated memory in GPU: %d MB\n",total_global_memory/(1024*1024));
             fprintf(stderr, "Failed to launch hipMalloc (part 1) (error code: %s)!\n", hipGetErrorString(err));
             exit(EXIT_FAILURE);
          }

  total_global_memory += (1+SW_links_per_cell+2*ARMS)*(NX*NY*NZ)*sizeof(byte);
  CudaSafeCall(hipMalloc( (void**)&dev_s, ARMS*NX*NY*NZ*sizeof(byte) ));
  CudaSafeCall(hipMalloc( (void**)&dev_active_bond, ARMS*NX*NY*NZ*sizeof(byte) ));
  CudaSafeCall(hipMalloc( (void**)&dev_nhb, NX*NY*NZ*sizeof(byte) ));
  CudaSafeCall(hipMalloc( (void**)&dev_delta, SW_links_per_cell*NX*NY*NZ*sizeof(byte) ));
 
          err = hipGetLastError();
          if (err != hipSuccess)
          {
             fprintf(stderr, "Allocated memory in GPU: %d MB\n",total_global_memory/(1024*1024));
             fprintf(stderr, "Failed to launch hipMalloc (part 2) (error code: %s)!\n", hipGetErrorString(err));
             exit(EXIT_FAILURE);
          }

  total_global_memory += (NX*NY*NZ)*(2*ARMS*sizeof(int) + sizeof(byte));
  CudaSafeCall(hipMalloc( (void**)&dev_label,ARMS*NX*NY*NZ*sizeof(int) ));
  CudaSafeCall(hipMalloc( (void**)&dev_prev_label,ARMS*NX*NY*NZ*sizeof(int) ));
  CudaSafeCall(hipMalloc( (void**)&dev_converges,NX*NY*NZ*sizeof(byte) ));

  total_global_memory += (5*N_RANDOM+SW_links_per_cell)*NX*NY*NZ*sizeof(float);
  CudaSafeCall(hipMalloc( (void**)&dev_Rnd, (NX*NY*NZ)*N_RANDOM*sizeof(float) ));
  CudaSafeCall(hipMalloc( (void**)&dev_Rnd_spin, (NX*NY*NZ)*N_RANDOM*sizeof(float) ));
  CudaSafeCall(hipMalloc( (void**)&dev_Rnd_cubeFlip, (NX*NY*NZ)*N_RANDOM*sizeof(float) ));
  CudaSafeCall(hipMalloc( (void**)&dev_Rnd_chess, (NX*NY*NZ)*N_RANDOM*sizeof(float) ));
  CudaSafeCall(hipMalloc( (void**)&dev_Rnd_cluster, (NX*NY*NZ)*SW_links_per_cell*sizeof(float) ));
  CudaSafeCall(hipMalloc( (void**)&dev_Rnd_update, (NX*NY*NZ)*N_RANDOM*sizeof(float) ));

          err = hipGetLastError();
          if (err != hipSuccess)
          {
             fprintf(stderr, "Allocated memory in GPU: %d MB\n",total_global_memory/(1024*1024));
             fprintf(stderr, "Failed to launch hipMalloc (part 3) (error code: %s)!\n", hipGetErrorString(err));
             exit(EXIT_FAILURE);
          }


  CudaSafeCall(hipMemcpy( dev_seed_array, seed_array, (NX*NY*NZ)*sizeof(uint), hipMemcpyHostToDevice ));

          err = hipGetLastError();
          if (err != hipSuccess)
          {
             fprintf(stderr, "Failed to launch hipMemcpy dev_seed_array (error code: %s)!\n", hipGetErrorString(err));
             exit(EXIT_FAILURE);
          }

  gpu_RNG_setup <<<nblocks,nthreads>>> (devStates, dev_seed_array);

          err = hipGetLastError();
          if (err != hipSuccess)
          {
             fprintf(stderr, "Failed to launch gpu_RNG_setup (error code: %s)!\n", hipGetErrorString(err));
             exit(EXIT_FAILURE);
          }

  
  gpu_RNG_generate <<<nblocks,nthreads>>> ( devStates, dev_Rnd, N_RANDOM);
  gpu_RNG_generate <<<nblocks,nthreads>>> ( devStates, dev_Rnd_spin, N_RANDOM);
  gpu_RNG_generate <<<nblocks,nthreads>>> ( devStates, dev_Rnd_cubeFlip, N_RANDOM);
  gpu_RNG_generate <<<nblocks,nthreads>>> ( devStates, dev_Rnd_chess, N_RANDOM);
  gpu_RNG_generate <<<nblocks,nthreads>>> ( devStates, dev_Rnd_cluster, SW_links_per_cell);
  gpu_RNG_generate <<<nblocks,nthreads>>> ( devStates, dev_Rnd_update, N_RANDOM);

          err = hipGetLastError();
          if (err != hipSuccess)
          {
             fprintf(stderr, "Failed to launch gpu_RNG_generate (error code: %s)!\n", hipGetErrorString(err));
             exit(EXIT_FAILURE);
          }
#endif

////
////  CHESSBOARD ALGORITHM
////

  if (flag_chessboard == 1){

      fprintf(logfile, "Starting Chessboard\n");

#ifdef CPU
      rand_Chessboard = (int*) calloc((NX*NY*NZ)/4,sizeof(int));

      initialize_Chessboard (NX, NY, NZ);

      for (int i =0; i<(NX*NY*NZ)/4; i++)
        rand_Chessboard[i] = (int) (NSOLUTIONS*mersenne());

      Chessboard_set_active_bonds_Typewriter(0, s, ARMS, NX*NY*NZ, Jeff , beta, rand_Chessboard, active_bond);
#endif

#ifdef GPU
      if (NX != NY || NX != NZ || NY != NZ ) { fprintf(stderr,"Error: can't build chessboard.\n\tThe system is not a cube.\n"); exit(1);} 
		//Maybe it could be solved for rectangles

      if ( NX % 4 != 0 ) { fprintf(stderr,"Error: can't build chessboard.\n\tUse NX muliply of 4.\n"); exit(1);} 

      hipError_t err = hipSuccess;

      total_global_memory += (1+3*EDGES)*(NX*NY*NZ/4)*sizeof(int);

      CudaSafeCall(hipMalloc( (void**)&devChessboard_vertex, EDGES*((NX*NY*NZ)/4)*2*sizeof(int) ));
      CudaSafeCall(hipMalloc( (void**)&devChessboard_edge, EDGES*((NX*NY*NZ)/4)*sizeof(int) ));
      CudaSafeCall(hipMalloc( (void**)&devChessboard_state, ((NX*NY*NZ)/4)*sizeof(int) ));

          err = hipGetLastError();
          if (err != hipSuccess)
          {
             fprintf(stderr, "Allocated memory in GPU: %d MB\n",total_global_memory/(1024*1024));
             fprintf(stderr, "Failed to launch hipMalloc devChessboard_* (error code: %s)!\n", hipGetErrorString(err));
             exit(EXIT_FAILURE);
          }

      gpu_set_cubes <<<nblocks,nthreads>>> (devChessboard_edge, devChessboard_vertex);
          err = hipGetLastError();
          if (err != hipSuccess)
          {
             fprintf(stderr, "Failed to launch gpu_set_cubes kernel (error code: %s)!\n", hipGetErrorString(err));
             exit(EXIT_FAILURE);
          }

      total_global_memory += EDGES*NSOLUTIONS*sizeof(int);
      CudaSafeCall(hipMalloc( (void**)&devSolutions, EDGES*NSOLUTIONS*sizeof(int)));

          err = hipGetLastError();
          if (err != hipSuccess)
          {
             fprintf(stderr, "Allocated memory in GPU: %d MB\n",total_global_memory/(1024*1024));
             fprintf(stderr, "Failed to launch hipMalloc devSolutions (error code: %s)!\n", hipGetErrorString(err));
             exit(EXIT_FAILURE);
          }

      int * cpuSolutions;
      cpuSolutions = (int*) malloc(EDGES*NSOLUTIONS*sizeof(int));

      setSolutions_Typewriter(cpuSolutions); // fill vector in host, then copy to device

      CudaSafeCall(hipMemcpy( devSolutions, cpuSolutions, EDGES*NSOLUTIONS*sizeof(int), hipMemcpyHostToDevice ));
          err = hipGetLastError();
          if (err != hipSuccess)
          {
             fprintf(stderr, "Failed to launch hipMemcpy devSolutions HostToDevice (error code: %s)!\n", hipGetErrorString(err));
             exit(EXIT_FAILURE);
          }
    
      gpu_Chessboard_start_cubes <<<nblocks,nthreads>>> (NX*NY*NZ, dev_Rnd_cubeFlip, devSolutions, dev_active_bond, 
          devChessboard_edge, devChessboard_vertex, devChessboard_state);

          err = hipGetLastError();
          if (err != hipSuccess)
          {
             fprintf(stderr, "Failed to launch gpu_Chessboard_start_cubes kernel (error code: %s)!\n", hipGetErrorString(err));
             exit(EXIT_FAILURE);
          }

      CudaSafeCall(hipMemcpy( active_bond, dev_active_bond, ARMS*NX*NY*NZ*sizeof(byte), hipMemcpyDeviceToHost ));

#endif
  } else {   //Chessboard algorithm is not active
     for (int i=0; i<ARMS*NX*NY*NZ; i++)
         active_bond[i] = 1;
  }
  
////
////  GENERATE INITIAL CONFIGURATION
////

  if (flag_config == 1){ /* Initial spins - read from file */
     iconfig = fopen(input_config,"r");
     if (iconfig == NULL){
        fprintf(stderr,"Can't open the initial configuration file\n");
        fprintf(stderr,"File named 'input_config' is expected\n");
        exit(1);
     }

     byte sp0, sp1, sp2, sp3, sp4, sp5;
     int ab0, ab1, ab2, ab3, ab4, ab5; //variables to read from file

     int i=0;
     while(fscanf(iconfig, "%d %d %d %d %d %d %d %d %d %d %d %d",&sp0,&sp1,&sp2,&sp3,&sp4,&sp5,
		&ab0,&ab1,&ab2,&ab3,&ab4,&ab5) == 12)  {

       s[0*NX*NY*NZ + i] = sp0;
       s[1*NX*NY*NZ + i] = sp1;
       s[2*NX*NY*NZ + i] = sp2;
       s[3*NX*NY*NZ + i] = sp3;
       s[4*NX*NY*NZ + i] = sp4;
       s[5*NX*NY*NZ + i] = sp5;

       active_bond[0*NX*NY*NZ + i] = ab0;
       active_bond[1*NX*NY*NZ + i] = ab1;
       active_bond[2*NX*NY*NZ + i] = ab2;
       active_bond[3*NX*NY*NZ + i] = ab3;
       active_bond[4*NX*NY*NZ + i] = ab4;
       active_bond[5*NX*NY*NZ + i] = ab5;

       for (int j=0; j<ARMS; j++){

         if (s[j*NX*NY*NZ + i] < 0 || s[j*NX*NY*NZ + i] > 5){
    	     fprintf(stderr,"Initial configuration file error\n");
             fprintf(stderr,"Unexpected value for spin (%d) appeared in line %d\n", s[j*NX*NY*NZ + i], i);
             fclose(iconfig);
             exit(1);
         }

         if (active_bond[j*NX*NY*NZ + i] < 0 || active_bond[j*NX*NY*NZ + i] > 1){
    	     fprintf(stderr,"Initial configuration file error\n");
             fprintf(stderr,"Unexpected value for active_bond (%d) appeared in line %d\n", active_bond[j*NX*NY*NZ + i], i);
             fclose(iconfig);
             exit(1);
         }
       } 

       i++;
       if (i > (NX*NY*NZ)){
         fprintf(stderr,"Initial configuration file error\n");
         fprintf(stderr,"There are more rows than cells in the file\n");
         fclose(iconfig);
         exit(1);
       }
  
     }
       
     if ( i < (NX*NY*NZ) ){
        fprintf(stderr,"Initial configuration file error\n");
        fprintf(stderr,"Unexpected ending. Can't fill the spin array\n");
        fclose(iconfig);
        exit(1);
     }
     fclose(iconfig);
     
     if ( flag_chessboard == 1 ){  //change cube states according to input active bonds
#ifdef GPU
     CudaSafeCall(hipMemcpy( dev_active_bond, active_bond, ARMS*NX*NY*NZ*sizeof(byte), hipMemcpyHostToDevice ));

  //   hipError_t err = hipSuccess;

     gpu_Chessboard_set_state <<<nblocks,nthreads>>> ( NX*NY*NZ, dev_active_bond, devChessboard_vertex, devChessboard_state ); 

          err = hipGetLastError();
          if (err != hipSuccess)
          {
             fprintf(stderr, "Failed to launch gpu_set_state kernel (error code: %s)!\n", hipGetErrorString(err));
             exit(EXIT_FAILURE);
          }

#endif
     }

  } else if (flag_config == 0) {   /* Initial spins - completely disordered */

    
    for (i=0; i<NX*NY*NZ; ++i)
      for (j=0; j<ARMS; ++j)
        s[j*NX*NY*NZ + i] = (byte)(q*mersenne());

  } else if (flag_config == 2) {  // Initial spins - completely ordered

    byte initial_spin = (byte)(q*mersenne());

    for (i=0; i<NX*NY*NZ; ++i)
      for (j=0; j<ARMS; ++j)
        s[j*NX*NY*NZ + i] = initial_spin;

  } else {
    fprintf(stderr,"Error: Wrong value for flag_config = %d\n",flag_config);
    fprintf(stderr,"Flag_config=0  Initial spins completely disordered\n");
    fprintf(stderr,"Flag_config=1  Initial spins read from file\n");
    fprintf(stderr,"Flag_config=2  Initial spins completely ordered (T=0)\n");
    exit(1);
  }

  int x,y,z;
  for (i=0; i < (NX*NY*NZ); ++i) {  //setting nhb

    x = i % NX;
    y = (i / NX)%NY;
    z = i / (NX*NY);

    int xup = (x+1)%NX + y*NX + z*NX*NY;
    int yup = x + ((y+1)%NY)*NX + z*NX*NY;
    int zup = x + y*NX + ((z+1)%NZ)*NX*NY;
    
    int xdn = (x-1+NX)%NX + y*NX + z*NX*NY;
    int ydn = x + ((y-1+NY)%NY)*NX + z*NX*NY;
    int zdn = x + y*NX + ((z-1+NZ)%NZ)*NX*NY;
    
    byte acc = 0;
    
    acc += active_bond[0*NX*NY*NZ + i] * delta(s[0*NX*NY*NZ + i],s[1*NX*NY*NZ + xup]);
    acc += active_bond[1*NX*NY*NZ + i] * delta(s[1*NX*NY*NZ + i],s[0*NX*NY*NZ + xdn]);
    
    acc += active_bond[2*NX*NY*NZ + i] * delta(s[2*NX*NY*NZ + i],s[3*NX*NY*NZ + yup]);
    acc += active_bond[3*NX*NY*NZ + i] * delta(s[3*NX*NY*NZ + i],s[2*NX*NY*NZ + ydn]);
    
    acc += active_bond[4*NX*NY*NZ + i] * delta(s[4*NX*NY*NZ + i],s[5*NX*NY*NZ + zup]);
    acc += active_bond[5*NX*NY*NZ + i] * delta(s[5*NX*NY*NZ + i],s[4*NX*NY*NZ + zdn]);
    
    nhb[i] = acc;
  }

  //setting distance and frequency
  if ( flag_distances == 0 ){
       x = 0 % NX;       // x, y, z for molecule 0
       y = (0 / NX)%NY;
       z = 0 / (NX*NY);

       for (int j=1; j < (NX*NY*NZ); ++j){   //calculate distances between cells and molecule 0
         int x2, y2, z2;
         float dist;
         int flag;

         x2 = j % NX;
         y2 = (j / NX)%NY;
         z2 = j / (NX*NY);

         dist = distance_PBC(x,y,z,x2,y2,z2);

         if (dist < R_cutoff) {
 
           flag = 1;
 
           for (int k=0; k<dist_index; ++k)
              if (dist == distance[k]){
                 flag = 0;
                 frequency[k] ++;
                 break;
              }

           if (flag){

              distance[dist_index] = dist;
              frequency[dist_index] = 1;
              dist_index ++;

              if (dist_index == num_distances){    //reallocation of distance and frequency
                 num_distances += int( cbrt( float(NX*NY*NZ) ) );
                 distance = (float*)realloc(distance,(num_distances)*sizeof(float));
                 frequency = (int*)realloc(frequency,(num_distances)*sizeof(int));
                 for (int l = dist_index; l<num_distances; ++l){
                     distance[l] = 0;
                     frequency[l] = 0;
                 }
              }
  
           }
         }  //end if dist>R_cutoff
       }  //end for j=i+1

       for(int l=0; l<dist_index; ++l){  // final result for full lattice
            frequency[l] /= 2;
            frequency[l] *= (NX*NY*NZ);
       }

      // writing into file
      char dist_file[200];
      FILE *odist;

      sprintf(dist_file,  "distances");
      odist = fopen(dist_file, "w");
      if(odist == NULL){
         fprintf(stderr,"Can't create distances file\n");
         exit(1);
      }

      fprintf(odist,"%d %d %d\n",NX,NY,NZ);
      for( int i=0; i<dist_index; ++i)
         fprintf(odist,"%f %d\n",distance[i],frequency[i]);

      fclose(odist);

  } else {     // reading distances from file

    char dist_file[200];
    FILE *idist;
    sprintf(dist_file,  "distances");
    idist = fopen(dist_file, "r");
    if(idist == NULL){
       fprintf(stderr,"Can't open distances file\n");
       exit(1);
    }

    int LX, LY, LZ;

    if (fscanf(idist,"%d %d %d", &LX, &LY, &LZ) != 3){
        fprintf(stderr,"Error: Can't read distances file\n");
        fclose(idist);
        exit(1);
    }


    if (LX != NX){
        fprintf(stderr,"Error: Distances file. X rank mismatch\n");
        fclose(idist);
        exit(1);
    }

    if ( LY != NY){
        fprintf(stderr,"Error: Distances file. Y rank mismatch\n");
        fclose(idist);
        exit(1);
    }

    if ( LZ!= NZ){
        fprintf(stderr,"Error: Distances file. Z rank mismatch\n");
        fclose(idist);
        exit(1);
    }

    dist_index=0;
    while(fscanf(idist,"%f %d", &distance[dist_index], &frequency[dist_index])==2){
       dist_index++;
       if (dist_index == num_distances){    //reallocation of distance and frequency
                 num_distances += int( cbrt( float(NX*NY*NZ) ) );
                 distance = (float*)realloc(distance,(num_distances)*sizeof(float));
                 frequency = (int*)realloc(frequency,(num_distances)*sizeof(int));
                 for (int l = dist_index; l<num_distances; ++l){
                     distance[l] = 0;
                     frequency[l] = 0;
                 }
       }
    }
  }

  num_distances = dist_index;
  distance = (float*)realloc(distance,num_distances*sizeof(float));  //free memory
  frequency = (int*)realloc(frequency,num_distances*sizeof(int));

// Cluster Probabilities (Wolff & Swendsen-Wang)
  pJ_hb = 1-exp(-beta*fabs(Jeff));   // Jeff = J_hb - P*v_hb where J_hb > 0 is attractive
  pJ_s = 1-exp(-beta*fabs(J_sig));   // J_sig > 0 is attractive
                                     // E_HB = -J_hb*N_HB - J_sig*N_sig

// Vectors for cluster statistics
  cluster_size_ = (int*) calloc(ARMS*NX*NY*NZ,sizeof(int));
  Xcoord = (byte*) calloc(NX,sizeof(byte));
  Ycoord = (byte*) calloc(NY,sizeof(byte));
  Zcoord = (byte*) calloc(NZ,sizeof(byte));
  
// initialize Wolff cluster algorithm variables
#ifdef WOLFF
  order = (int*)calloc(ARMS,sizeof(int));
  for ( i=0; i<ARMS; i++ )
      order[i] = i;

  neigh_arm = (int*)calloc(ARMS,sizeof(int));
  neigh_arm[0] = 1;
  neigh_arm[1] = 0;
  neigh_arm[2] = 3;
  neigh_arm[3] = 2;
  neigh_arm[4] = 5;
  neigh_arm[5] = 4;

  is_cluster = (bool*) calloc(ARMS*NX*NY*NZ,sizeof(bool));
  for(i =0; i<ARMS*NX*NY*NZ; i++)
    is_cluster[i] = false;

  neighbor = (int*) calloc(ARMS*NX*NY*NZ,sizeof(int));  
  for(i=0; i<ARMS*NX*NY*NZ; i++){
     int arm = i/(NX*NY*NZ);
     int cell = i%(NX*NY*NZ);

     int x = cell % NX;
     int y = (cell/NX)%NY;
     int z = cell/(NX*NY);

    switch(arm){
      case 0: // forward X
        neighbor[i] = (x+1)%NX + y*NX + z*NX*NY;
        break;
      case 1: // backward X
        neighbor[i] = (x-1+NX)%NX + y*NX + z*NX*NY;
        break;
      case 2: // forward Y
        neighbor[i] = x + ((y+1)%NY)*NX + z*NX*NY;
        break;
      case 3: // backward Y
        neighbor[i] = x + ((y - 1 + NY)%NY)*NX + z*NX*NY;
        break;
      case 4: // forward Z
        neighbor[i] = x + y*NX + ((z+1)%NZ)*NX*NY;
        break;
      case 5: // backward Z
        neighbor[i] = x + y*NX + ((z-1+NZ)%NZ)*NX*NY;
        break;
    }
     
  }

  cluster_size = 0;
#endif

#ifdef SWENDSEN_WANG
  cpu_label = (int*) calloc(ARMS*NX*NY*NZ,sizeof(int));  
  //prev_label = (int*) calloc(ARMS*NX*NY*NZ,sizeof(int));

  cpu_delta = (byte*) calloc(SW_links_per_cell*NX*NY*NZ,sizeof(byte));
  converges = (byte*) calloc(NX*NY*NZ,sizeof(byte));
  //device SW variables are allocated elsewhere and initialize at the beginning of each step with a GPU kernel
  //check if cuda Atomic function could be used for checking convergence instead of this method

#endif

#ifdef cpu_SWENDSEN_WANG

  SW_bonded = (int**) calloc(ARMS*NX*NY*NZ,sizeof(int*));
  for (int i=0; i<ARMS*NX*NY*NZ; i++)
    SW_bonded[i] = (int*) calloc(ARMS,sizeof(int));
   
  //Hoshen Kopelman labels
  cpu_SW_L = (int*) calloc (ARMS*NX*NY*NZ,sizeof(int));
  cpu_SW_N = (int*) calloc (ARMS*NX*NY*NZ,sizeof(int));
  HK_label = (int*) calloc (ARMS*NX*NY*NZ,sizeof(int));
  
 // cpu_SW_L2 = (int*) calloc (ARMS*NX*NY*NZ,sizeof(int));
  
 // cpu_SW_L3 = (int*) calloc (ARMS*NX*NY*NZ,sizeof(int));
 // visited = (int*) calloc (ARMS*NX*NY*NZ,sizeof(int));
  
  neighbor_spin = (int*) calloc (ARMS,sizeof(int));
  SW_new_spin = (int*) calloc (ARMS*NX*NY*NZ,sizeof(int));

#endif

////
////  ALLOCATE AND INITIALZE CORRELATORS
////

// Observable M = fraction of spins at the most populated state

  int total_correlators = 1;    // change to NX*NY*NZ to recover correlation previous to 3.1
  if ( flag_correlation == 1 ){

   Corr = (struct correlator*) malloc(sizeof(struct correlator)*total_correlators);

   for(i = 0; i < total_correlators; i ++) {
      Corr[i].shift = (double **)malloc(sizeof(double *)*numcorrelators);
      for(j = 0; j < numcorrelators; j ++)
          Corr[i].shift[j] = (double *)calloc(p,sizeof(double));
       
      Corr[i].correlation = (double **)malloc(sizeof(double *)*numcorrelators);
      for(j = 0; j < numcorrelators; j ++)
          Corr[i].correlation[j] = (double *)calloc(p,sizeof(double));
       
      Corr[i].ncorrelation = (long int **)malloc(sizeof(long int *)*numcorrelators);
      for(j = 0; j < numcorrelators; j ++)
          Corr[i].ncorrelation[j] = (long int *)calloc(p,sizeof(long int));
        
      Corr[i].accumulator = (double *)calloc(numcorrelators,sizeof(double));
      Corr[i].naccumulator = (int *)calloc(numcorrelators,sizeof(int));
      Corr[i].insertindex = (int *)calloc(numcorrelators,sizeof(int));
      Corr[i].t = (double *)calloc(numcorrelators*p,sizeof(double));
      Corr[i].f = (double *)calloc(numcorrelators*p,sizeof(double));
    }

   initialize(total_correlators,Corr);

   histo_sigma = (int*) calloc(q,sizeof(int));
   
  }

////
////  FIRST CALCULATION OF VDW ENERGY
////

  calculate_energy_vdW ( &energy_vdW );
  
////
////  COPY SYSTEM FROM HOST TO DEVICE
////

#ifdef GPU  
  CudaSafeCall(hipMemcpy( dev_s, s, ARMS*NX*NY*NZ*sizeof(byte), hipMemcpyHostToDevice ));

  CudaSafeCall(hipMemcpy( dev_active_bond, active_bond, ARMS*NX*NY*NZ*sizeof(byte), hipMemcpyHostToDevice ));

  CudaSafeCall(hipMemcpy( dev_nhb, nhb, NX*NY*NZ*sizeof(byte), hipMemcpyHostToDevice ));
#endif

  offset = 0;  //Metropolis random number
  loop = 0;
  offset_chess = 0; //Chessboard random number
  loop_chess = 0;
  offset_update = 0; //Swendsen-Wang random number
  loop_update = 0;

 /* {  // write first config to compare to config.out //
        FILE * oconfig_time;
        char output_config_time[200];
        sprintf(output_config_time,"config0");
        oconfig_time = fopen(output_config_time,"w");
        for (int k=0; k<NX*NY*NZ; ++k){
           fprintf(oconfig_time,"%d %d %d %d %d %d ",s[0*NX*NY*NZ + k],s[1*NX*NY*NZ + k],s[2*NX*NY*NZ + k],s[3*NX*NY*NZ + k],s[4*NX*NY*NZ + k],s[5*NX*NY*NZ + k]);
           fprintf(oconfig_time,"%d %d %d %d %d %d\n",active_bond[0*NX*NY*NZ + k],active_bond[1*NX*NY*NZ + k],active_bond[2*NX*NY*NZ + k],active_bond[3*NX*NY*NZ + k],active_bond[4*NX*NY*NZ + k],active_bond[5*NX*NY*NZ + k]);
        }
        fclose(oconfig_time);
  } */

  max_cluster_size = 0;
  max_cluster_size2 = 0;
  SW_cluster_size = (int*) calloc(ARMS*NX*NY*NZ,sizeof(int));

  FILE *logsize = fopen("cluster.out", "w");
  if(logsize == NULL){
    fprintf(stderr,"Can't open input file\n");
    exit(1);
  }
  
////
////  EQULIIBRATION STEPS
////
  
  /* the update/equilibrate loop */
  for (i=0; i<n_equilibrium; ++i){
  
   //  printf("time=%d\n",i);

 /*    clock_t start, end;
     double cpu_time_used;
    
     if(i%2000 == 0)
        start = clock();*/

    MonteCarlo_Step(i, loop, offset, loop_chess, offset_chess, nblocks, nthreads, order, logfile, SW_cluster_size, logsize);

  /*   if((i+1)%2000 == 0){
         end = clock(); 
         cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
         printf("Equil. step %d, lasted %f\n",i,cpu_time_used);
     }*/

  }

  max_cluster_size = 0;
  max_cluster_size2 = 0;
  
////
////  PRODUCTION RUN
////
          
  /* and the update/measure loop */
  for (i=0; i<n_loops; ++i) {

 //   if(i % (n_loops/10) == 0)
 //       fprintf(logfile,"%.1f%% elapsed\n",10*i/(n_loops/10.));
    

  /*   clock_t start, end;
     double cpu_time_used;
    
     if(i%2000 == 0)
        start = clock(); */
 
 //printf("time=%d\n",i);

    MonteCarlo_Step(i+n_equilibrium, loop, offset, loop_chess, offset_chess, nblocks, nthreads, order, logfile, SW_cluster_size,logsize);

   /*  if((i+1)%2000 == 0){
         end = clock(); 
         cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
         printf("Sampl. step %d, lasted %f\n",i,cpu_time_used); 

     }*/
    
    if (i % N_MEASURE == 0){
#ifdef GPU
      CudaSafeCall(hipMemcpy( s, dev_s, ARMS*NX*NY*NZ*sizeof(byte), hipMemcpyDeviceToHost ));
      CudaSafeCall(hipMemcpy( active_bond, dev_active_bond, ARMS*NX*NY*NZ*sizeof(byte), hipMemcpyDeviceToHost ));
      CudaSafeCall(hipMemcpy( nhb, dev_nhb, NX*NY*NZ*sizeof(byte), hipMemcpyDeviceToHost ));
#endif

#ifdef SWENDSEN_WANG 
     CudaSafeCall(hipMemcpy( cpu_label, dev_label, ARMS*NX*NY*NZ*sizeof(int), hipMemcpyDeviceToHost ));
     print_cluster_statistics(i,logsize, cpu_label);
#endif

#ifdef cpu_SWENDSEN_WANG
      print_cluster_statistics(i,logsize, HK_label);
#endif

      measure(i,odata);
    }

    if ( flag_correlation == 1 ){
#ifdef GPU
      CudaSafeCall(hipMemcpy( s, dev_s, ARMS*NX*NY*NZ*sizeof(byte), hipMemcpyDeviceToHost ));
#endif
      for (int k=0;k<total_correlators; k++){
//        double sigma = (double) (s[0*NX*NY*NZ + k]+s[1*NX*NY*NZ + k]+s[2*NX*NY*NZ + k]+s[3*NX*NY*NZ + k]+s[4*NX*NY*NZ + k]+s[5*NX*NY*NZ + k]);
//        sigma /= 6.0;
        for (int l=0; l<q; l++)
           histo_sigma[l] = 0;

        for (int l=0; l<ARMS*NX*NY*NZ; l++){

           if ( s[l] < 0 || s[l] >= q ) { 
              fprintf(stderr,"ERROR while correlation estimate: s[%d]=%d out of range\n",
                            l,s[l]);
              exit(1);
           }
           histo_sigma[s[l]] ++;
        }

        double max_mass = -2E10;
        for (int l=0; l<q; l++)
          if (histo_sigma[l] > max_mass) max_mass = histo_sigma[l];

        max_mass /= (double) (ARMS*NX*NY*NZ);

        add(Corr,k,max_mass,0);
      }
    }
  } //end of main loop

  fclose(odata);
  fclose(logsize);
  
////
////  STORE DATA AND FINAL CALCULATIONS
////

  M /= n_loops*1./N_MEASURE;
  M2 /= n_loops*1./N_MEASURE;
  E /= n_loops*1./N_MEASURE;
  E2 /= n_loops*1./N_MEASURE;

  double dM = sqrt(M2 - M*M);
  double dE = sqrt(E2 - E*E);

  fprintf(logfile,"Magnetization:\t%1.6lf\t+/-\t%1.6lf\n", M, dM);
  fprintf(logfile,"Energy:\t\t%1.6lf\t+/-\t%1.6lf\n", E, dE);
  
  /* Final spins - write output file */
  oconfig = fopen(output_config,"w");
#ifdef GPU
  CudaSafeCall(hipMemcpy( s, dev_s, ARMS*NX*NY*NZ*sizeof(byte), hipMemcpyDeviceToHost ));
  CudaSafeCall(hipMemcpy( active_bond, dev_active_bond, ARMS*NX*NY*NZ*sizeof(byte), hipMemcpyDeviceToHost ));
#endif
  for (i=0; i<NX*NY*NZ; ++i){
      fprintf(oconfig,"%d %d %d %d %d %d ",s[0*NX*NY*NZ + i],s[1*NX*NY*NZ + i],s[2*NX*NY*NZ + i],s[3*NX*NY*NZ + i],s[4*NX*NY*NZ + i],s[5*NX*NY*NZ + i]);
      fprintf(oconfig,"%d %d %d %d %d %d\n",active_bond[0*NX*NY*NZ + i],active_bond[1*NX*NY*NZ + i],active_bond[2*NX*NY*NZ + i],active_bond[3*NX*NY*NZ + i],active_bond[4*NX*NY*NZ + i],active_bond[5*NX*NY*NZ + i]);
  }
  fclose(oconfig);

  /* Correlation evaluation */
  if (flag_correlation == 1) {
    evaluate(Corr,total_correlators,1);

    char filename[100];
    sprintf(filename,"correlation.dat");
    printCorr(Corr,filename,total_correlators);
  }

  fprintf(logfile,"Max_SW_scans = %d\n",max_SW_scans);
  fprintf(logfile," ** simulation done\n");
  fclose(logfile);
  
  return 0;
}

float distance_PBC(int x1, int y1, int z1, int x2, int y2, int z2){
   int x_PBC, y_PBC, z_PBC;

   x_PBC = abs(x1 - x2);
   if ( x_PBC > NX/2 ) x_PBC = NX - x_PBC;

   y_PBC = abs(y1 - y2);
   if ( y_PBC > NY/2 ) y_PBC = NY - y_PBC;

   z_PBC = abs(z1 - z2);
   if ( z_PBC > NZ/2 ) z_PBC = NZ - z_PBC;

   return sqrt(x_PBC*x_PBC + y_PBC*y_PBC + z_PBC*z_PBC);

}

////
////  MONTE CARLO STEP
////	Update eta variables (if chessboard enabled)
////	Update r_cell
////	Update sigma variables (Metropolis, SW, or Wolff)
////

void MonteCarlo_Step(int time, int &loop, int &offset, int &loop_chess, int &offset_chess, int nblocks, int nthreads, int *order, FILE* logfile, int *SW_cluster_size, FILE *logsize){

    if (flag_chessboard == 1){

#ifdef CPU
    	for (int i =0; i<(NX*NY*NZ)/4; i++)
   	   rand_Chessboard[i] = (int) (9*mersenne());  //warning: I haven't checked if mersenne() returns [0,1) or (0,1]
 
	Chessboard_set_active_bonds_Typewriter(1,s, ARMS, NX*NY*NZ, Jeff , beta, rand_Chessboard, active_bond);
#endif

#ifdef GPU
        offset_chess = ((NX*NY*NZ)/4)*loop_chess;  

        hipError_t err2 = hipSuccess;

        gpu_Chessboard_set_active_bonds <<<nblocks,nthreads>>> (dev_s, NX*NY*NZ, Jeff, beta,
                                       dev_Rnd_cubeFlip, dev_Rnd_chess, offset_chess, devSolutions, dev_active_bond,
                                       devChessboard_edge, devChessboard_vertex, devChessboard_state);
          err2 = hipGetLastError();
          if (err2 != hipSuccess)
          {
             fprintf(stderr, "Failed to launch gpu_Chessboard_set_active_bonds kernel (error code: %s)!\n", hipGetErrorString(err2));
             exit(EXIT_FAILURE);
          }

        loop_chess ++;

        if(loop_chess == 4*N_RANDOM) { 
          loop_chess = 0;
          gpu_RNG_generate <<<nblocks,nthreads>>> (devStates, dev_Rnd_cubeFlip, N_RANDOM);
          gpu_RNG_generate <<<nblocks,nthreads>>> (devStates, dev_Rnd_chess, N_RANDOM);
        } 

        CudaSafeCall(hipMemcpy( active_bond, dev_active_bond,ARMS*NX*NY*NZ*sizeof(byte), hipMemcpyDeviceToHost ));
#endif
    }

    volume_step(&r_cell, &energy_vdW, &V);

#ifdef gpu_METROPOLIS
#ifdef GPU
    hipError_t err3 = hipSuccess;
    for (int j=0; j<NMETROPOLIS; j++){

      for(int k=0; k < 2*ARMS; ++k) {

        byte spin = (byte) ARMS*mersenne();
        int indices = (int) 2*mersenne();

        offset = (NX*NY*NZ)*loop;

        gpu_update <<<nblocks,nthreads>>> (dev_s,dev_active_bond,dev_nhb,dev_Rnd,
                                           dev_Rnd_spin,indices,P, T, V, offset, spin,flag_chessboard);
          err3 = hipGetLastError();
          if (err3 != hipSuccess)
          {
             fprintf(stderr, "Failed to launch gpu_update kernel (error code: %s)!\n", hipGetErrorString(err3));
             exit(EXIT_FAILURE);
          }

        ++loop;

        if(loop == N_RANDOM) {
          loop = 0;
          gpu_RNG_generate <<<nblocks,nthreads>>> ( devStates, dev_Rnd, N_RANDOM);
          gpu_RNG_generate <<<nblocks,nthreads>>> ( devStates, dev_Rnd_spin, N_RANDOM);
        }
      }
    }
#endif
#endif

#ifdef cpu_METROPOLIS
    for (int j=0; j<NMETROPOLIS; j++)
       cpu_update();
       
  #ifdef GPU
        CudaSafeCall(hipMemcpy( dev_s, s, ARMS*NX*NY*NZ*sizeof(byte), hipMemcpyHostToDevice ));
  #endif
#endif

#ifdef WOLFF
    if (NCLUSTER > 0){
#ifdef GPU
        CudaSafeCall(hipMemcpy( s, dev_s, ARMS*NX*NY*NZ*sizeof(byte), hipMemcpyDeviceToHost )); 
        CudaSafeCall(hipMemcpy( active_bond, dev_active_bond,ARMS*NX*NY*NZ*sizeof(byte), hipMemcpyDeviceToHost ));
#endif
        for (int j=0; j<NCLUSTER; j++)
           cluster_step();

     //   fprintf(logsize,"%d %d %f %d\n",time,tmp_max_cluster_size,average_cluster_size,num_clusters);
#ifdef GPU
        CudaSafeCall(hipMemcpy( dev_s, s, ARMS*NX*NY*NZ*sizeof(byte), hipMemcpyHostToDevice ));
#endif
    }
#endif  //end ifdef Wolff

#ifdef SWENDSEN_WANG
#ifdef GPU
    hipError_t err4;
    if (NCLUSTER > 0){

        for (int j=0; j<NCLUSTER; j++){

            tmp_max_cluster_size = 0;

//            printf("Inicio paso SW\n");

            gpu_initialize_cluster_variables <<<nblocks,nthreads>>> (dev_delta, dev_label, dev_prev_label);
                err4 = hipGetLastError();
                if (err4 != hipSuccess)
                {
                   fprintf(stderr, "Failed to launch gpu_initialize_cluster_variables kernel (error code: %s)!\n", hipGetErrorString(err4));
                   exit(EXIT_FAILURE);
                }

       /*     CudaSafeCall(hipMemcpy( active_bond, dev_active_bond, ARMS*NX*NY*NZ*sizeof(byte), hipMemcpyDeviceToHost ));
            fprintf(logfile,"Initial ab[]\n");
            for (int jj=0; jj<ARMS*NX*NY*NZ; jj++)
              fprintf(logfile,"\tI ab[%d] = %d\n",jj,active_bond[jj]);

*/
            gpu_create_cluster <<<nblocks,nthreads>>> (dev_Rnd_cluster, dev_delta, dev_s, dev_active_bond, pJ_hb, pJ_s, Jeff);
                err4 = hipGetLastError();
                if (err4 != hipSuccess)
                {
                   fprintf(stderr, "Failed to launch gpu_create_cluster kernel (error code: %s)!\n", hipGetErrorString(err4));
                   exit(EXIT_FAILURE);
                }

            cluster_converges = false;
            int loop_counter=0;

            while ( ! cluster_converges ){

                loop_counter ++;

    //            fprintf(logfile,"Step %d Scanning\n",loop_counter); 

                gpu_cluster_scanning_covalent <<<nblocks,nthreads>>> (dev_delta, dev_label);
                   err4 = hipGetLastError();
                   if (err4 != hipSuccess)
                   {
                      fprintf(stderr, "Failed to launch gpu_cluster_scanning kernel (error code: %s)!\n",
                                      hipGetErrorString(err4));
                      exit(EXIT_FAILURE);
                   }

      /*          CudaSafeCall(hipMemcpy( cpu_label, dev_label, ARMS*NX*NY*NZ*sizeof(int), hipMemcpyDeviceToHost ));
                fprintf(logfile,"Scanning_Covalent Step %d Label[]\n",loop_counter);
                for (int jj=0; jj<ARMS*NX*NY*NZ; jj++)
                   fprintf(logfile,"\tScanning_Covalent %d Label[%d] = %d\n",loop_counter,jj,cpu_label[jj]);
        */    

                gpu_cluster_scanning_sigma <<<nblocks,nthreads>>> (dev_delta, dev_label);
                   err4 = hipGetLastError();
                   if (err4 != hipSuccess)
                   {
                      fprintf(stderr, "Failed to launch gpu_cluster_scanning kernel (error code: %s)!\n",
                                      hipGetErrorString(err4));
                      exit(EXIT_FAILURE);
                   }

          /*      CudaSafeCall(hipMemcpy( cpu_label, dev_label, ARMS*NX*NY*NZ*sizeof(int), hipMemcpyDeviceToHost ));
                fprintf(logfile,"Scanning_Sigma Step %d Label[]\n",loop_counter);
                for (int jj=0; jj<ARMS*NX*NY*NZ; jj++)
                   fprintf(logfile,"\tScanning_Sigma %d Label[%d] = %d\n",loop_counter,jj,cpu_label[jj]);
            */

                for (int ii=0; ii<ARMS; ii++){

                   gpu_cluster_analysis <<<nblocks,nthreads>>> (dev_label, ii);
                      err4 = hipGetLastError();
                      if (err4 != hipSuccess)
                      {
                         fprintf(stderr, "Failed to launch gpu_cluster_analysis kernel (error code: %s)!\n",
                                         hipGetErrorString(err4));
                         exit(EXIT_FAILURE);
                      }

                }

              /*  CudaSafeCall(hipMemcpy( cpu_label, dev_label, ARMS*NX*NY*NZ*sizeof(int), hipMemcpyDeviceToHost ));
                fprintf(logfile,"Analysis Step %d Label[]\n",loop_counter);
                for (int jj=0; jj<ARMS*NX*NY*NZ; jj++)
                   fprintf(logfile,"\tAnalysis %d Label[%d] = %d\n",loop_counter,jj,cpu_label[jj]);
            */

                /// CONVERGENCE TEST 
                gpu_convergence_test <<<nblocks,nthreads>>> (dev_label, dev_prev_label, dev_converges);
                   err4 = hipGetLastError();
                   if (err4 != hipSuccess)
                   {
                      fprintf(stderr, "Failed to launch gpu_convergence_test kernel (error code: %s)!\n",
                                      hipGetErrorString(err4));
                      exit(EXIT_FAILURE);
                   }
                CudaSafeCall(hipMemcpy( converges, dev_converges, NX*NY*NZ*sizeof(byte), hipMemcpyDeviceToHost ));

              //  fprintf(logfile,"Test Convergence Step %d\n",loop_counter);
                cluster_converges = true;
                for (int ii=0; ii<NX*NY*NZ; ii++)
                    if ( converges[ii] != 0 ) {
                //       fprintf(logfile,"\tCell %d has not converged. Conv=%d\n",ii,converges[ii]);
                       cluster_converges = false;
                       break;
                    }

            }   

        //    fprintf(logfile,"Cluster has converged\n");
            loop_counter --;
            if (loop_counter > max_SW_scans) max_SW_scans = loop_counter;

            offset_update = (NX*NY*NZ)*loop_update;

            gpu_update_cluster <<<nblocks,nthreads>>> (dev_label, dev_s, offset_update, dev_Rnd_update);
                err4 = hipGetLastError();
                if (err4 != hipSuccess)
                {
                   fprintf(stderr, "Failed to launch gpu_update_cluster kernel (error code: %s)!\n", hipGetErrorString(err4));
                   exit(EXIT_FAILURE);
                }


          /*  CudaSafeCall(hipMemcpy( cpu_label, dev_label, ARMS*NX*NY*NZ*sizeof(int), hipMemcpyDeviceToHost ));
	    fprintf(logfile,"Final Label[]\n");
            for (int jj=0; jj<ARMS*NX*NY*NZ; jj++)
		fprintf(logfile,"\tF Label[%d] = %d\n",jj,cpu_label[jj]);

            CudaSafeCall(hipMemcpy( s, dev_s, ARMS*NX*NY*NZ*sizeof(byte), hipMemcpyDeviceToHost ));
            fprintf(logfile,"Final s[]\n");
            for (int jj=0; jj<ARMS*NX*NY*NZ; jj++)
              fprintf(logfile,"\tF s[%d] = %d\n",jj,s[jj]);
*/
            loop_update ++;

            if (loop_update == N_RANDOM){
                loop_update = 0;
                gpu_RNG_generate <<<nblocks,nthreads>>> (devStates, dev_Rnd_update, N_RANDOM);
            }

            gpu_RNG_generate <<<nblocks,nthreads>>> (devStates, dev_Rnd_cluster, SW_links_per_cell);
                err4 = hipGetLastError();
                if (err4 != hipSuccess)
                {
                   fprintf(stderr, "Failed to launch gpu_RNG_generate SW cluster kernel (error code: %s)!\n", hipGetErrorString(err4));
                   exit(EXIT_FAILURE);
                }

       //     printf("End SW step\n");

        }       
    }
        
#endif
#endif //end ifdef SWENDSEN_WANG

#ifdef cpu_SWENDSEN_WANG
#ifdef GPU
        CudaSafeCall(hipMemcpy( s, dev_s, ARMS*NX*NY*NZ*sizeof(byte), hipMemcpyDeviceToHost )); 
        CudaSafeCall(hipMemcpy( active_bond, dev_active_bond,ARMS*NX*NY*NZ*sizeof(byte), hipMemcpyDeviceToHost ));
#endif

   if (NCLUSTER > 0){

        for (int j=0; j<NCLUSTER; j++){

          cpu_SW_initialize();
          
          for ( int k=NX*NY*NZ; k<ARMS*NX*NY*NZ; k++){  //create matrix of bonds
          
             cpu_SW_neighboring_spins(k);  // set vector neighbor_spin[]
             
             // covalent bonds
             if ( neighbor_spin[0] < k ){  //avoid double attempt create same bond
             
               if ( Jeff > 0 ){ 
               
                  if ( active_bond[k] != active_bond[neighbor_spin[0]] ){
                     fprintf(stderr,"cpu_SW detecta un error en active_bond on el calculo del vecino.\n");
                     exit(1);
                  }
               
                  if ( active_bond[k] ){
                        if ( s[k]  == s[neighbor_spin[0]] ){
                            if ( mersenne() < pJ_hb )
                                SW_bonded[k][0] = 1; 
                        }
                      
                  }
               } else {
                  if ( s[k]  != s[neighbor_spin[0]] ){
                        if ( mersenne() < pJ_hb )
                            SW_bonded[k][0] = 1; 
                  }      
               }
               
             }
               
             // cooperative bonds
             for ( int kk=1; kk<ARMS; kk++){
                if ( neighbor_spin[kk] < k ){ //avoid double attempt create same bond
                  if ( s[k] == s[neighbor_spin[kk]] ){
                        if ( mersenne() < pJ_s )
                            SW_bonded[k][kk] = 1; 
                  }             
                }
               
             }
    
          } // end matrix of bonds 
       
          
          //Hoshen Kopelman Algorithm
          int cpu_SW_converges = 0;
          while ( cpu_SW_converges == 0 ){
          
             cpu_SW_converges = 1;
              
             for ( int k=NX*NY*NZ; k<ARMS*NX*NY*NZ; k++){
    
                 cpu_SW_neighboring_spins(k);
               
                 int nbonds = 0;
                 int min_label = 2*ARMS*NX*NY*NZ;
                 for ( int kk=0 ; kk<ARMS; kk++){    // estimate min_label
               
                    if ( neighbor_spin[kk] < k )
                      if ( SW_bonded[k][kk] == 1 ){
                         nbonds ++;
                       
                         if ( cpu_SW_L[neighbor_spin[kk]] < min_label )
                            min_label = cpu_SW_L[neighbor_spin[kk]];
                      }
                 }
               
                 if ( nbonds >= 1 )
                     cpu_SW_L[k] = min_label;
                 
              
                 if ( nbonds > 1 ){
              
                     int new_N = -2*ARMS*NX*NY*NZ;

                     for ( int kk=0 ; kk<ARMS; kk++){  //find max N among bonded spins
               
                       if ( neighbor_spin[kk] < k )
                         if ( SW_bonded[k][kk] == 1 ){  
                             if ( new_N < cpu_SW_N[cpu_SW_L[neighbor_spin[kk]]] )
                                new_N = cpu_SW_N[cpu_SW_L[neighbor_spin[kk]]];
                                
                         }
                    
                     }
                  
                     for ( int kk=0 ; kk<ARMS; kk++){
               
                       if ( neighbor_spin[kk] < k )
                         if ( SW_bonded[k][kk] == 1 ){
                         
                           if ( cpu_SW_N[ cpu_SW_L[neighbor_spin[kk]] ] != new_N){ 
                              cpu_SW_converges = 0;
                              cpu_SW_N[ cpu_SW_L[neighbor_spin[kk]] ] = new_N;   
                           }
                           
                         }
                        
                     }
                 }             
             }
         
          }  //end Hoshen Kopelman
          
          cpu_SW_update(HK_label);

        }
   }
   
   
#ifdef GPU
        CudaSafeCall(hipMemcpy( dev_s, s, ARMS*NX*NY*NZ*sizeof(byte), hipMemcpyHostToDevice ));
#endif

#endif //end ifdef cpu_SWENDSEN_WANG

}// end subroutine Monte Carlo Step


////
////  VDW VOLUME STEP
////

void volume_step(double * r_cell, double * energy, double * volume)
{

    double trial_r_cell;
    double volume_step_epsilon = 0.01;
    
    do {
        trial_r_cell = (*r_cell) + volume_step_epsilon* 2 *( uniformDoubleRand() - 0.5 );
    } while ( trial_r_cell < LJ_InfiniteBarrier );
    
    double trial_volume = trial_r_cell*trial_r_cell*trial_r_cell * NX*NY*NZ;
    double dV = trial_volume - (*volume);         
    
    double trial_energy = 0, tmp, attractive, repulsive, shift;
    
    // shift: correction on the potential to avoid step at the cutoff
    tmp = 1.0/(R_cutoff);
    tmp = tmp * tmp * tmp;
    attractive = tmp * tmp;
    repulsive = attractive * attractive;
    shift = (repulsive - attractive);

    for ( int i = 0 ; i < num_distances ; ++i )  
      if ( trial_r_cell * distance[i] < R_cutoff ){
          tmp = 1.0/( trial_r_cell * distance[i] );
          tmp = tmp * tmp * tmp;
          attractive = tmp * tmp;
          repulsive = attractive * attractive;
          trial_energy += frequency[i] * (repulsive - attractive - shift);
      }

    
    double dE = ( trial_energy - (*energy) ) - NX*NY*NZ * T * log( trial_volume/(*volume) );

    if ( uniformDoubleRand() < exp( - beta *( dE + P* dV )) )
    {
        * r_cell = trial_r_cell;
        * volume = trial_volume;
        * energy = trial_energy;
    }

}


void calculate_energy_vdW (double * energy){

    double tmp, attractive, repulsive, ret=0, shift;
    
    // shift: correction on the potential to avoid step at the cutoff
    tmp = 1.0/(R_cutoff);
    tmp = tmp * tmp * tmp;
    attractive = tmp * tmp;
    repulsive = attractive * attractive;
    shift = (repulsive - attractive);

    for ( int i = 0 ; i < num_distances ; ++i )  //energy of one cell
						 //energy of the lattice when multiplying by frequency[i]
       if ( r_cell * distance[i] < R_cutoff ){
           tmp = 1.0/( r_cell * distance[i] );
           tmp = tmp * tmp * tmp;
           attractive = tmp * tmp;
           repulsive = attractive * attractive;
           ret += frequency[i] * (repulsive - attractive - shift);
       }
     * energy = ret;
}

////
////  CPU METROPOLIS ALGORITHM
////

void cpu_update() 
{
   
  int this_site, cell, arm, neigh_cell, neighbor_spin,x,y,z;
  int new_spin, old_spin;
  float dE;
  for (int j=0; j < ARMS*NX*NY*NZ; ++j) {
  
    this_site = (int) (mersenne()*ARMS*NX*NY*NZ);
    
    old_spin = s[this_site];
    new_spin = (int) (q*mersenne());
    
    if ( old_spin == new_spin ) continue;
    
    cell = this_site%(NX*NY*NZ);
    arm = this_site/(NX*NY*NZ);
    
    x = cell % NX;
    y = (cell / NX) % NY;
    z = cell / (NX*NY);
    
    switch (arm){
   
      case 0:
        neigh_cell = (x+1)%NX + y*NX + z*NX*NY;
        neighbor_spin = 1*NX*NY*NZ+neigh_cell;
        break;
        
      case 1:
        neigh_cell = (x-1+NX)%NX + y*NX + z*NX*NY;
        neighbor_spin = 0*NX*NY*NZ+neigh_cell;
        break;
        
      case 2:
        neigh_cell = x + ((y+1)%NY)*NX + z*NX*NY;
        neighbor_spin = 3*NX*NY*NZ+neigh_cell;
        break;
        
      case 3:
        neigh_cell = x + ((y-1+NY)%NY)*NX + z*NX*NY;
        neighbor_spin = 2*NX*NY*NZ+neigh_cell;
        break;
        
      case 4:
        neigh_cell = x + y*NX + ((z+1)%NZ)*NX*NZ;
        neighbor_spin = 5*NX*NY*NZ+neigh_cell;
        break;
        
      case 5:
        neigh_cell = x + y*NX + ((z-1+NZ)%NZ)*NX*NY;
        neighbor_spin = 4*NX*NY*NZ+neigh_cell;
        break; 
        
      default:
        break;  
    }
    
    dE = - Jeff * active_bond[this_site] * (delta(new_spin,s[neighbor_spin]) -  delta(old_spin,s[neighbor_spin]));
    
    for ( int k=0; k<ARMS; k++ ){
       if ( k == arm ) continue; 
       neighbor_spin = k*NX*NY*NZ + cell;
       dE -= J_sig * ( delta(new_spin,s[neighbor_spin]) - delta(old_spin,s[neighbor_spin]) );
    }
    
    if ( dE <= 0 )
       s[this_site] = new_spin;
    else if ( mersenne() < exp(-beta*dE) )
       s[this_site] = new_spin;
      
  }   

}

////
////  GPU METROPOLIS ALGORITHM
////

__global__ void gpu_update (
  byte * dev_s,
  byte * dev_active_bond,
  byte * dev_nhb,
  float * dev_Rnd,
  float * dev_Rnd_spin,
  int indices,
  float P,
  float T,
  float V,
  int offset,
  byte spin_index,
  int flag_chessboard)
{
  /* 1D grid of 1D blocks */
    uint tid = blockIdx.x*blockDim.x + threadIdx.x;
	
    uint x = tid % NX;
    uint y = (tid/NX) % NY;
    uint z = tid/(NX*NY);

    // water model algorithm
      
    if(spin_index == 0 || spin_index == 1) { // update Y-Z planes
      if(indices && x%2)
        return;
      else if(!indices && !(x%2))
        return;
    } else if(spin_index == 2 || spin_index == 3) { // update X-Z planes
      if(indices && y%2)
        return;
      else if(!indices && !(y%2))
        return;
    } else if(spin_index == 4 || spin_index == 5) { // update X-Y planes
      if(indices && z%2)
        return;
      else if(!indices && !(z%2))
        return;
    }
        
    /// attempt to coalesced reads and writes
    // coalesced read
    uint neighbor;
    byte neig_spin;
    
    //  NO warp divergence in this block, spin_index is the same for all threads
    switch(spin_index){
      case 0: // forward X
        neighbor = (x+1)%NX + y*NX + z*NX*NY;
        neig_spin = dev_s[1*NX*NY*NZ + neighbor];
        break;
      case 1: // backward X
        neighbor = (x-1+NX)%NX + y*NX + z*NX*NY;
        neig_spin = dev_s[0*NX*NY*NZ + neighbor];
        break;
      case 2: // forward Y
        neighbor = x + ((y+1)%NY)*NX + z*NX*NY;
        // nearly coalesced
        neig_spin = dev_s[3*NX*NY*NZ + neighbor];
        break;
      case 3: // backward Y
        neighbor = x + ((y - 1 + NY)%NY)*NX + z*NX*NY;
        neig_spin = dev_s[2*NX*NY*NZ + neighbor];
        break;
      case 4: // forward Z
        neighbor = x + y*NX + ((z+1)%NZ)*NX*NY;
        neig_spin = dev_s[5*NX*NY*NZ + neighbor];
        break;
      case 5: // backward Z
        neighbor = x + y*NX + ((z-1+NZ)%NZ)*NX*NY;
        neig_spin = dev_s[4*NX*NY*NZ + neighbor];
        break;
    }
    
    byte old_spin = dev_s[spin_index*NX*NY*NZ + tid];
    
    // attempt spin change
    //  byte new_spin = (old_spin + 1 + (byte)((q-1)*dev_Rnd_spin[tid + offset])) % q;
      byte new_spin = (byte) (q*dev_Rnd_spin[tid + offset]);
      new_spin %= q;    // avoid event new_spin = 6

    // compute new energy
    int8_t deltaNHB = (int8_t)(delta(new_spin,neig_spin) - delta(old_spin,neig_spin));
    
    byte nhbi = dev_nhb[tid];
    byte nhbj = dev_nhb[neighbor];
    
    float deltaE = 0;
    if ( flag_chessboard == 1 ){
       deltaE += delta(new_spin,neig_spin) * dev_active_bond[spin_index*NX*NY*NZ + tid];
       deltaE -= delta(old_spin,neig_spin) * dev_active_bond[spin_index*NX*NY*NZ + tid];
    }else {
       deltaE += (theta(nhbi+deltaNHB,NHBMAX) * theta(nhbj+deltaNHB,NHBMAX)) * delta(new_spin,neig_spin);
       deltaE -= (theta(nhbi,NHBMAX)          * theta(nhbj,NHBMAX))          * delta(old_spin,neig_spin);
    }
        
    deltaE *= (-J_hb + P*v_hb);
    
    for (uint i=0; i < ARMS; ++i)
    {
	byte spin = dev_s[i*NX*NY*NZ + tid];
        // coalesced reads
        deltaE += -J_sig * (delta(new_spin,spin) - delta(old_spin,spin));
    }
    
    // avoid double counting of same spin
    deltaE -= J_sig;    //warning: if (new_spin == old_spin) deltaE = -J_sig, but it should be deltaE = 0
			//This mistake has no effect in simulation because it doesn't change resulting dev_s[] and dev_nhb[]    

    // metropolis acceptance rule
    if (dev_Rnd[tid + offset] > expf(-deltaE/T)) {
      new_spin = old_spin;
      deltaNHB = 0;
    }
        
    dev_s[spin_index*NX*NY*NZ + tid] = new_spin;
    dev_nhb[tid] += deltaNHB;
    dev_nhb[neighbor] += deltaNHB;
}

////
////  MEASUREMENT SUBROUTINE
////

void measure(int iter,FILE *f)
{
  /* measure energy and magnetization, and print out
   */
  int i,j,k,x,y,z,xup,yup,zup,neighbor,neig_spin;
  double e,h,m,v;
  double n_hb = 0;
  
  /* sum over the neighbour sites - typewriter fashion */
  e = m = 0;

  int count_nhb[NX*NY*NZ];
  for (i=0; i < (NX*NY*NZ); ++i)
    count_nhb[i] = 0;
  
  for (i=0; i < (NX*NY*NZ); ++i) {
    x = i % NX;
    y = (i / NX)%NY;
    z = i / (NX*NY);
    xup = (x+1)%NX + y*NX + z*NX*NY;
    yup = x + ((y+1)%NY)*NX + z*NX*NY;
    zup = x + y*NX + ((z+1)%NZ)*NX*NY;

    if (flag_chessboard == 1){

      // forward X
      neighbor = xup;
      neig_spin = s[1*NX*NY*NZ + neighbor];
      if ( delta(s[0*NX*NY*NZ + i], neig_spin) ){
         count_nhb[i] += active_bond[0*NX*NY*NZ + i];
         count_nhb[neighbor] += active_bond[1*NX*NY*NZ + neighbor];
      }

      // forward Y
      neighbor = yup;
      neig_spin = s[3*NX*NY*NZ + neighbor];
      if ( delta(s[2*NX*NY*NZ + i], neig_spin) ){
         count_nhb[i] += active_bond[2*NX*NY*NZ + i];
         count_nhb[neighbor] += active_bond[3*NX*NY*NZ + neighbor];
      }

      // forward Z
      neighbor = zup;
      neig_spin = s[5*NX*NY*NZ + neighbor];
      if ( delta(s[4*NX*NY*NZ + i], neig_spin) ){
         count_nhb[i] += active_bond[4*NX*NY*NZ + i];
         count_nhb[neighbor] += active_bond[5*NX*NY*NZ + neighbor];
      }

    }  // end count nhb with chessboard
    else {

      // forward X
      neighbor = xup;
      neig_spin = s[1*NX*NY*NZ + neighbor];
      if(delta(s[0*NX*NY*NZ + i], neig_spin)) {
        n_hb ++;
        if(count_nhb[i] < NHBMAX && count_nhb[neighbor] < NHBMAX) {
          count_nhb[i] ++;
          count_nhb[neighbor] ++;
        }
      }
    
      // forward Y
      neighbor = yup;  
      neig_spin = s[3*NX*NY*NZ + neighbor];
      if(delta(s[2*NX*NY*NZ + i], neig_spin)) {
        n_hb ++;
        if(count_nhb[i] < NHBMAX && count_nhb[neighbor] < NHBMAX) {
          count_nhb[i] ++;
          count_nhb[neighbor] ++;
        }
      }
      
      // forward Z
      neighbor = zup;
      neig_spin = s[5*NX*NY*NZ + neighbor];
      if(delta(s[4*NX*NY*NZ + i], neig_spin)) {
        n_hb ++;
        if(count_nhb[i] < NHBMAX && count_nhb[neighbor] < NHBMAX) {
          count_nhb[i] ++;
          count_nhb[neighbor] ++;
        }
   
      }
    }  // end count nhb without chessboard

    for(j=0; j < ARMS-1; ++j)
      for(k=j+1; k < ARMS; ++k) 
        m += delta(s[j*NX*NY*NZ + i], s[k*NX*NY*NZ + i]);
	
  } // end loop over cells

 // check count_nhb
 /* 
 for (i=0; i < (NX*NY*NZ); ++i) {
        if (count_nhb[i] < 0 || count_nhb[i] > NHBMAX ){
              printf("ERROR! nhb(%d)=%d\n",i, count_nhb[i]);
              exit(1);
        }
  }*/
  
  // check the distribution of sigma states across the systems
  /*
  if(iter%(NLOOPS/10)==0) {
    printf("iter %d\n",iter);
    printf("sigmas\n");
    for(j=0; j < q; ++j)
      printf("%d: %1.2f%%\n",j,sigma_dist[j]*100./(NX*NY*NZ*q));
    printf("hbs\n");
    for(j=0; j < 1+ARMS; ++j)
      printf("%d: %1.2f%%\n",j,nhb_dist[j]*100./(NX*NY*NZ));
    
    
    int check_nhb=0;
    for(i=0; i < NX*NY*NZ; ++i) {
      if(nhb[i] < 0)
        printf("warning! nhb(%d)=%d\n",i, nhb[i]);
      check_nhb += nhb[i];
    }  
    if(check_nhb != (int)(2*n_hb))
      printf("warning! check nhb: %.0f %d\n",2*n_hb,check_nhb);
  }*/

  calculate_energy_vdW ( &e );
  
  v = NX*NY*NZ*r_cell*r_cell*r_cell;

  n_hb = 0;  //now it is the total hb taking into account the restriction NHBMAX = 4
  for (i=0; i < (NX*NY*NZ); ++i){
    v += 0.5*count_nhb[i] * v_hb;
    e += 0.5*count_nhb[i] * (-J_hb);
    n_hb += 0.5*count_nhb[i];
  }

  e += (-J_sig)*m;  
  h = e + P*v;  // enthalpy

  m /= 1.*NX*NY*NZ*(ARMS*(ARMS-1)/2.);
  
  E += e;
  E2 += e*e;
  M += m;
  M2 += m*m;
  
  fprintf(f,"%d %1.7e %1.7e %1.7e %1.7e %1.7e %1.7e\n", iter, e, h, v, n_hb, m, r_cell);
}

/////////////////  SAVE CONFIGS TO CALCULATE CORRELATIONS   /////////////////////

/*
void set_logaritmic_tiimes()
{
        int max_power = 10;
        int num_of_times = 9*max_power + 1;
        int logaritmic_times[num_of_times];
        int power = 1;
        int k = 0;
        for(int i=0; i<max_power; i++){
                for(int j=1; j<=9; j++){
                        logaritmic_times[k] = j*power;
                        k++;
                }
                power *= 10;
        }
        logaritmic_times[k] = power;
}


bool is_logaritmic_time (int time)
{
        bool ret = false;
        int i = 0;
        while (!ret && i<num_of_times){
                if(time == logaritmic_times[i]) ret = true;
                i++;
        }
        return ret;
}
*/

////
////  WOLF CLUSTER ALGORITHM
////

void cluster_step()
{
   int cumm_cluster_size = 0;

   tmp_max_cluster_size = 0;
   num_clusters = 0;
   int cs;
   while (cumm_cluster_size < ARMS*NX*NY*NZ){
    //     cumm_cluster_size += cluster_poke();
         cs = cluster_poke();
         cumm_cluster_size += cs;
         num_clusters ++;
         if ( cs > tmp_max_cluster_size ) tmp_max_cluster_size = cs;
   }

   average_cluster_size = (float) cumm_cluster_size / (float) num_clusters;
  
}

int site (int arm, int cell)
{ 
   if (arm*NX*NY*NZ + cell >= NX*NY*NZ*6){
	printf("ERROR (1) SITE \n");
        printf("arm=%d/6 ; cell=%d/%d; res=%d/%d\n",arm, cell,NX*NY*NZ,arm*NX*NY*NZ + cell,NX*NY*NZ*6);
        exit(-1);
   } else if (arm*NX*NY*NZ + cell < 0){
	printf("ERROR (2) SITE \n");
        exit(-1);
   }

   return arm*NX*NY*NZ + cell; 
}

int cluster_poke ()
{

  int cell = (int) NX*NY*NZ*mersenne();
  int arm = (int) q*mersenne();
  int this_spin = site(arm,cell);

  byte old_spin = s[this_spin];

  shuffle();

  byte new_spin = (byte) ( (q-1)*mersenne() + 1 );  // integer between 1 and q-1
        // I will rotate the cluster in update_cluster by adding new_spin to old_spin

  if ( new_spin <=0 || new_spin >= q){
	fprintf(stderr,"ERROR Cluster: new_spin out of range\n ns=%d\n",new_spin);
        exit(-1);	
  }

  cluster_size = add_to_cluster( arm, cell );

  int neigh_spin = site(neigh_arm[arm], neighbor[this_spin]);
  if ( ! is_cluster[ neigh_spin ] ){

      if ( Jeff > 0 )
      {
         if (  ( active_bond[ neigh_spin ] == ACTIVE ) &&
               ( s[ this_spin ] == s[ neigh_spin ] ) &&
               ( mersenne() < pJ_hb )   ){

                cluster_size += add_to_cluster( neigh_arm[arm] , neighbor[this_spin] );
                update_cluster (neighbor[this_spin],new_spin);
         }
      }
      else if ( Jeff < 0 )
      {
         if (s[ this_spin ] != s[ neigh_spin ] &&
             mersenne() < pJ_hb ){
                cluster_size += add_to_cluster( neigh_arm[arm] , neighbor[this_spin] );
                update_cluster (neighbor[this_spin],new_spin);
         }          
      }

  }

  update_cluster(cell, new_spin);

  return cluster_size;
}

void shuffle()
{

    int temp, pos;
    for ( int i = 0 ; i < q ; i++ )
    {
        temp = order[i];
        pos = (int) q*mersenne();
        order[i] = order[pos];
        order[pos] = temp;
    }
}

int add_to_cluster( int arm, int cell ){

   int this_spin = site(arm,cell);  //seed spin of this (sub)cluster
 
   is_cluster[this_spin] = true;
   int local_size = 1;

   for (int i=0; i<6; i++){
      int next_arm = order[i];
      int next_spin = site(next_arm,cell);

      if ( ! is_cluster[next_spin] && s[next_spin] == s[this_spin] && mersenne() < pJ_s){

         local_size += add_to_cluster(next_arm, cell);

         if ( active_bond[ next_spin ] == NON_ACTIVE ) continue;

         int neigh_spin = site(neigh_arm[next_arm] , neighbor[next_spin] );
         if ( ! is_cluster[ neigh_spin ] ){

               if ( Jeff > 0 )
               {
                  if (s[ next_spin ] == s[ neigh_spin ] &&
                      mersenne() < pJ_hb )

                         local_size += add_to_cluster( neigh_arm[next_arm] , neighbor[next_spin] );
               }
               else if ( Jeff < 0 )
               {
                  if (s [next_spin ] != s[ neigh_spin ] &&
                      mersenne() < pJ_hb )
                         local_size += add_to_cluster( neigh_arm[next_arm] , neighbor[next_spin] );         
               }

         }
         
      }

   }

   return local_size;
}

void update_cluster( int cell, int new_spin ){

  for (int i=0; i<6; i++){
    int arm = order[i];
    int this_spin = site(arm,cell);

    if ( is_cluster[this_spin] )
    {
       s[this_spin] = ( s[this_spin] + new_spin ) % q;
       is_cluster[this_spin] = false;

       int neigh_spin = site(neigh_arm[arm] , neighbor[this_spin]);
       if ( is_cluster[ neigh_spin ] ){
          update_cluster(neighbor[this_spin],new_spin);
       }     
    }

  }

}

////
////  GPU SWENDSEN WANG CLUSTER ALGORITHM
////

// Follows algorithm described by Y. Komura and Y. Okabe in https://doi.org/10.1016/j.cpc.2012.01.017

__global__ void gpu_initialize_cluster_variables (byte *dev_delta, int *dev_label, int *dev_prev_label)
{

    int tid = blockIdx.x*blockDim.x + threadIdx.x;

    int sp;

    for ( int i=0; i<SW_links_per_cell; i++ )
      dev_delta[i*NX*NY*NZ+tid] = 0;

    for ( int i=0; i<ARMS; i++ ){
      sp = i*NX*NY*NZ+tid;
      dev_label[sp] = sp;
      dev_prev_label[sp] = sp;
    }

}

__global__ void gpu_create_cluster ( float * dev_Rnd_cluster, byte * dev_delta, byte * dev_s,
   byte * dev_active_bond, float pJ_hb, float pJ_s, float Jeff)
{

    int tid = blockIdx.x*blockDim.x + threadIdx.x;

    uint x = tid % NX;
    uint y = (tid/NX) % NY;
    uint z = tid/(NX*NY);

    uint xup = (x+1)%NX + y*NX + z*NX*NY;
    uint yup = x + ((y+1)%NY)*NX + z*NX*NY;
    uint zup =  x + y*NX + ((z+1)%NZ)*NX*NZ;


    if ( Jeff > 0 ) //no thread divergence, Jeff is the same for all threads
    {

      if ( dev_active_bond[0*NX*NY*NZ+tid] && delta(dev_s[0*NX*NY*NZ+tid],dev_s[1*NX*NY*NZ+xup]) &&
           dev_Rnd_cluster[0*NX*NY*NZ+tid] < pJ_hb )
        dev_delta[0*NX*NY*NZ+tid] = 1;

      if ( dev_active_bond[2*NX*NY*NZ+tid] && delta(dev_s[2*NX*NY*NZ+tid],dev_s[3*NX*NY*NZ+yup]) && 
           dev_Rnd_cluster[1*NX*NY*NZ+tid] < pJ_hb )
        dev_delta[1*NX*NY*NZ+tid] = 1;

      if ( dev_active_bond[4*NX*NY*NZ+tid] && delta(dev_s[4*NX*NY*NZ+tid],dev_s[5*NX*NY*NZ+zup]) &&
           dev_Rnd_cluster[2*NX*NY*NZ+tid] < pJ_hb )
        dev_delta[2*NX*NY*NZ+tid] = 1;

    } else if ( Jeff < 0) 
    {

      if ( dev_s[0*NX*NY*NZ+tid] != dev_s[1*NX*NY*NZ+xup] && dev_Rnd_cluster[0*NX*NY*NZ+tid] < pJ_hb )
        dev_delta[0*NX*NY*NZ+tid] = 1;

      if ( dev_s[2*NX*NY*NZ+tid] != dev_s[3*NX*NY*NZ+yup] && dev_Rnd_cluster[1*NX*NY*NZ+tid] < pJ_hb )
        dev_delta[1*NX*NY*NZ+tid] = 1;

      if ( dev_s[4*NX*NY*NZ+tid] != dev_s[5*NX*NY*NZ+zup] && dev_Rnd_cluster[2*NX*NY*NZ+tid] < pJ_hb )
        dev_delta[2*NX*NY*NZ+tid] = 1;

    }

    int link_counter = 3;
    for ( int ii=0; ii<ARMS-1; ii ++){
       for ( int jj=ii+1; jj<ARMS; jj++){

          if ( delta(dev_s[ii*NX*NY*NZ+tid],dev_s[jj*NX*NY*NZ+tid]) && dev_Rnd_cluster[link_counter*NX*NY*NZ+tid] < pJ_s )
             dev_delta[link_counter*NX*NY*NZ+tid] = 1;

          link_counter ++;
       }
    }

}
 
__global__ void gpu_cluster_scanning_covalent (byte * dev_delta, int * dev_label)
{

    int tid = blockIdx.x*blockDim.x + threadIdx.x;

    uint x = tid % NX;
    uint y = (tid/NX) % NY;
    uint z = tid/(NX*NY);

    uint xup = (x+1)%NX + y*NX + z*NX*NY;
    uint yup = x + ((y+1)%NY)*NX + z*NX*NY;
    uint zup =  x + y*NX + ((z+1)%NZ)*NX*NY;

    //scan xup
    if( dev_delta[0*NX*NY*NZ+tid] ){
      if ( dev_label[0*NX*NY*NZ+tid] < dev_label[1*NX*NY*NZ+xup] )
         dev_label[1*NX*NY*NZ+xup] = dev_label[0*NX*NY*NZ+tid];
      else
         dev_label[0*NX*NY*NZ+tid] = dev_label[1*NX*NY*NZ+xup];
    }

    //scan yup
    if( dev_delta[1*NX*NY*NZ+tid] ){
      if ( dev_label[2*NX*NY*NZ+tid] < dev_label[3*NX*NY*NZ+yup] )
         dev_label[3*NX*NY*NZ+yup] = dev_label[2*NX*NY*NZ+tid];
      else
         dev_label[2*NX*NY*NZ+tid] = dev_label[3*NX*NY*NZ+yup];
    }

    //scan zup
    if( dev_delta[2*NX*NY*NZ+tid] ){
      if ( dev_label[4*NX*NY*NZ+tid] < dev_label[5*NX*NY*NZ+zup] )
         dev_label[5*NX*NY*NZ+zup] = dev_label[4*NX*NY*NZ+tid];
      else
         dev_label[4*NX*NY*NZ+tid] = dev_label[5*NX*NY*NZ+zup];
    }

}

__global__ void gpu_cluster_scanning_sigma (byte * dev_delta, int * dev_label)
{
    int tid = blockIdx.x*blockDim.x + threadIdx.x;

    //scan sigma bonds
    int link_counter = 3;
    for ( int ii=0; ii<ARMS-1; ii ++){
       for ( int jj=ii+1; jj<ARMS; jj++){

          if ( dev_delta[link_counter*NX*NY*NZ+tid] ){

             if ( dev_label[ii*NX*NY*NZ+tid] < dev_label[jj*NX*NY*NZ+tid])
                  dev_label[jj*NX*NY*NZ+tid] = dev_label[ii*NX*NY*NZ+tid];
             else
                  dev_label[ii*NX*NY*NZ+tid] = dev_label[jj*NX*NY*NZ+tid];

          }

          link_counter ++;
       }
    }

}

// This implementation suffers from race conditions. It should not be a problem as they
// will eventually be solved correctly in subsequently calls of the scanning and analysis kernels.
// Splitting analysis into six calls from sp=0 to sp=5 minimize the problem.
__global__ void gpu_cluster_analysis(int *dev_label, int sp)
{
    int tid = blockIdx.x*blockDim.x + threadIdx.x;

    int idx = sp*NX*NY*NZ+tid;

     if ( dev_label[dev_label[idx]] != dev_label[idx] )
        dev_label[idx] = dev_label[dev_label[idx]];

}

__global__ void gpu_convergence_test(int *dev_label, int * dev_prev_label, byte * dev_converges)
{
    int tid = blockIdx.x*blockDim.x + threadIdx.x;

    int sp;
    dev_converges[tid]=0;
    for (int k=0; k<ARMS; k++){
      sp = k*NX*NY*NZ+tid;
      if ( dev_prev_label[sp] != dev_label[sp] ) dev_converges[tid] = 1;
      dev_prev_label[sp] = dev_label[sp];  //update for the next step in the loop
    }

}

__global__ void gpu_update_cluster(int * dev_label, byte * dev_s, int offset_update, float * dev_Rnd_update)
{
    int tid = blockIdx.x*blockDim.x + threadIdx.x;

    int sp;
    for ( int arm=0; arm<ARMS; arm++ ){
       sp = arm*NX*NY*NZ+tid;
       dev_s[sp] += (byte) (q*dev_Rnd_update[dev_label[sp]+offset_update]);
       dev_s[sp] = dev_s[sp]%q;  
    }

}

////
////  SEQUENTIAL CPU SWENDSEN WANG CLUSTER ALGORITHM
////

void cpu_SW_initialize(){

   // matrix of bonds
   for (int i=0; i<ARMS*NX*NY*NZ; i++){
      for (int j=0; j<ARMS; j++)
        SW_bonded[i][j] = 0; 
   }
   
   //Hoshen Kopelman labels
   for (int i=0; i<ARMS*NX*NY*NZ; i++){
      cpu_SW_L[i] = i;
      cpu_SW_N[i] = -i;
 /*     cpu_SW_L2[i] = i;
      cpu_SW_L3[i] = -1;
      visited[i] = 0;*/
   }

}

void cpu_SW_neighboring_spins(int i){

   int cell = i % (NX*NY*NZ);
   int arm = i / (NX*NY*NZ);
   
   int x = cell % NX;
   int y = (cell/NX) % NY;
   int z = cell/(NX*NY);
   
   int neigh_cell;
   
   switch (arm){
   
      case 0:
        neigh_cell = (x+1)%NX + y*NX + z*NX*NY;
        neighbor_spin[0] = 1*NX*NY*NZ+neigh_cell;
        break;
        
      case 1:
        neigh_cell = (x-1+NX)%NX + y*NX + z*NX*NY;
        neighbor_spin[0] = 0*NX*NY*NZ+neigh_cell;
        break;
        
      case 2:
        neigh_cell = x + ((y+1)%NY)*NX + z*NX*NY;
        neighbor_spin[0] = 3*NX*NY*NZ+neigh_cell;
        break;
        
      case 3:
        neigh_cell = x + ((y-1+NY)%NY)*NX + z*NX*NY;
        neighbor_spin[0] = 2*NX*NY*NZ+neigh_cell;
        break;
        
      case 4:
        neigh_cell = x + y*NX + ((z+1)%NZ)*NX*NZ;
        neighbor_spin[0] = 5*NX*NY*NZ+neigh_cell;
        break;
        
      case 5:
        neigh_cell = x + y*NX + ((z-1+NZ)%NZ)*NX*NY;
        neighbor_spin[0] = 4*NX*NY*NZ+neigh_cell;
        break; 
        
      default:
        break;  
   }
   
   int idx = 1;
   for ( int j=0; j<ARMS; j++ ){
   
       if ( j == arm ) continue;
       
       neighbor_spin[idx] = j*NX*NY*NZ+cell;
  
       idx ++;
   }
   
}


void cpu_SW_update(int * HK_label){

   for ( int i=0; i<ARMS*NX*NY*NZ ; i++)
      SW_new_spin[i] = -1;

   int cluster_label;
   for ( int i=0; i<ARMS*NX*NY*NZ ; i++){
   
      cluster_label = i;
      
      
      do{   //N is not enough to label the clusters. This loop looks for N[i] = -i
         cluster_label = - cpu_SW_N[cpu_SW_L[cluster_label]];
      }while( cluster_label !=  - cpu_SW_N[cpu_SW_L[cluster_label]] );  
      
      cpu_SW_N[cpu_SW_L[i]] = - cluster_label;  // update N to its correct value. 
                                       // Shortcut for future appearences of the same label
      
      if ( - cpu_SW_N[cluster_label] != cpu_SW_L[cluster_label] ){
   //      fprintf(stderr,"Time %d. ERROR Seq. Swendsen Wang: An error in Hoshen Kopelman algorithm occured\n",time);
         fprintf(stderr,"L[%d] = %d , L[%d] = %d\n",i,cpu_SW_L[i],cluster_label,cpu_SW_L[cluster_label]);
         fprintf(stderr,"N[%d] = %d , N[%d] = %d\n",cpu_SW_L[i],cpu_SW_N[cpu_SW_L[i]],cluster_label,cpu_SW_N[cluster_label]);
         exit(1);
      }
      
      if ( SW_new_spin[cluster_label] == -1 )
         SW_new_spin[cluster_label] = (int) (q*mersenne());
         
      if ( SW_new_spin[cluster_label] < 0 || SW_new_spin[cluster_label] > 5 ){
         fprintf(stderr,"Error: SW new spin mal calculado\n");
         exit(1);
      }
      
      s[i] = (byte) ( (s[i] + SW_new_spin[cluster_label]) % q );
      
      if ( s[i] < 0 || s[i] > 5 ){
         fprintf(stderr,"Error: SW new spin mal calculado\n");
         exit(1);
      }
      
      HK_label[i] = cluster_label;
      
   }

}

////
//// CLUSTER ANALYSIS SUBROUTINES
////

void print_cluster_statistics (int time, FILE * logsize, int * label){

   for ( int i=0; i<ARMS*NX*NY*NZ ; i++)  //loop over labels
      cluster_size_[i] = 0;
   
   for ( int i=0; i<ARMS*NX*NY*NZ ; i++){ //loop over sigma_ij variables
   
      cluster_size_[label[i]] ++;
      
   }
   
   int largest_size=0, second_size=0,num_clusters=0,num_percolating_clusters = 0;
   float average=0, average2=0;
   
   for ( int i=0; i<ARMS*NX*NY*NZ ; i++){ //loop over labels
   
       if ( cluster_size_[i] == 0 ) continue;
       
       num_clusters ++;
       average += cluster_size_[i];
       average2 += cluster_size_[i]*cluster_size_[i];
       
       if ( cluster_size_[i] > largest_size ){
          second_size = largest_size;
          largest_size = cluster_size_[i];
       } else if ( cluster_size_[i] > second_size ){
          second_size = cluster_size_[i];
       }
             
       // minimum size of a percolating cluster is 2L (chain from 0 to L, including up and dn spins)
       if ( cluster_size_[i] >= 2*NX || cluster_size_[i] >= 2*NY || cluster_size_[i] >= 2*NZ )
           num_percolating_clusters += is_percolating_cluster(label,i);  // returns 1 if percolates, 0 otherwise
       
   }
      
   average /= (float) num_clusters;
   average2 /= (float) num_clusters;
   
   average2 -= average*average;
    
   fprintf(logsize,"%d %d %d %d %f %f %d\n",time,largest_size,second_size,num_percolating_clusters,
                                         average,average2,num_clusters);  
   

}

int is_percolating_cluster(int * label, int id){
 
   for ( int i=0; i<NX; i++ )
       Xcoord[i] = 0;
   for ( int i=0; i<NY; i++ )
       Ycoord[i] = 0;
   for ( int i=0; i<NZ; i++ )
       Zcoord[i] = 0;
     
   int counter=0;
   for ( int i=0; i<ARMS*NX*NY*NZ; i++ ){  // loop over sigma_ij variables

         if ( label[i] == id ){
               
            int cell = i%(NX*NY*NZ);   
            int x = cell % NX;
            int y = (cell/NX) % NY;
            int z = cell/(NX*NY);
            
            Xcoord[x] = 1;
            Ycoord[y] = 1;
            Zcoord[z] = 1;
            
            counter ++;
         
         }  
         
         if ( counter == cluster_size_[id] ) //the whole cluster has been ckecked
            break;
            
   }
   
   int flag_x=1,flag_y=1,flag_z=1;
   
   for ( int i=0; i<NX; i++ )
      if  ( Xcoord[i] == 0 ){
         flag_x = 0;
         break;
      }
      
   for ( int i=0; i<NY; i++ )
      if  ( Ycoord[i] == 0 ){
         flag_y = 0;
         break;
      }
      
   for ( int i=0; i<NZ; i++ )
      if  ( Zcoord[i] == 0 ){
         flag_z = 0;
         break;
      }
   
   if ( flag_x == 1 || flag_y == 1 || flag_z==1 )
      return 1;
   else
      return 0;

}

///////////////////////////////////////////////////////////////////////////////////////////////

////
//// UTILITES
////

void compute_indices (int * indicesA,int * indicesB)
{
  int i,x,y,z,iA=0,iB=0;
  
  for (i=0; i < NX*NY*NZ; ++i) {
    
    x = i % NX;
    y = (i / NX) % NY;
    z = i / (NX*NY);

    if (z % 2) {
      if (y % 2) { 
        if (x % 2)
          indicesA[iA++] = i;
        else
          indicesB[iB++] = i;
      }
      else {
        if (x % 2)
          indicesB[iB++] = i;
        else
          indicesA[iA++] = i;
      }
    }
    else {
      if (y % 2) { 
        if (x % 2)
          indicesB[iB++] = i;
        else
          indicesA[iA++] = i;
      }
      else {
        if (x % 2)
          indicesA[iA++] = i;
        else
          indicesB[iB++] = i;
      }
    }
  }
}

////
////  GPU RANDOM NUMBER GENERATOR
////

__global__ void gpu_RNG_setup ( 
	hiprandState * state, 
	uint * seed)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    //hiprand_init( (seed << 20) + id, 0, 0, &state[id]);
    hiprand_init( seed[id], 0, 0, &state[id]);
}

__global__ void gpu_RNG_generate ( 
	hiprandState* globalState, 
	float * Rnd,
        int n_rand) 
{
    int ind = blockIdx.x * blockDim.x + threadIdx.x;
    
    hiprandState localState = globalState[ind];
    
    while(ind < (NX*NY*NZ) *n_rand) {
	    
	   // __syncthreads();
	    
	    Rnd[ind] = hiprand_uniform( &localState );   //returns a random number within the range (0, 1]
	    
	    ind += blockDim.x*gridDim.x;
    }
    
    ind = blockIdx.x * blockDim.x + threadIdx.x;
    
    globalState[ind] = localState; 
}


inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
	if ( hipSuccess != err )
	{
		fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
		file, line, hipGetErrorString( err ) );
		exit( -1 );
	}
#endif
	
	return;
}

inline void __cudaCheckError( const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
	hipError_t err = hipGetLastError();
	if ( hipSuccess != err )
	{
		fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
		file, line, hipGetErrorString( err ) );
		exit( -1 );
	}
	 
	// More careful checking. However, this will affect performance.
	// Comment away if needed.
//	err = hipDeviceSynchronize();
//	if( hipSuccess != err )
//	{
//		fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
//		file, line, hipGetErrorString( err ) );
//		exit( -1 );
//	}
#endif
	 
	return;
}




//////////////////////// FAST CUDA RANDOM GENERATOR ////////////////////////////



// S1, S2, S3, and M are all constants, z is the inner state  
__device__
static uint TausStep(uint &z, int S1, int S2, int S3, uint M) {  
  uint b=(((z << S1) ^ z) >> S2); 
  return z = (((z & M) << S3) ^ b);  
} 

__device__
// A and C are constants 
static uint LCGStep(uint &z, uint A, uint C) {  
  return z=(A*z+C);  
} 


__device__
static float HybridTaus(uint& z1, uint& z2, uint& z3, uint& z4) {  
  // Combined period is lcm(p1,p2,p3,p4)~ 2^121
  float randval;

  //return 2.3283064365387e-10f*LCGStep(z4, 1664525, 1013904223UL);

  //do { 
   randval = 2.3283064365387e-10f * (          // Periods  
    TausStep(z1, 13, 19, 12, 4294967294UL) ^  // p1=2^31-1  
    TausStep(z2, 2, 25, 4, 4294967288UL) ^    // p2=2^30-1  
    TausStep(z3, 3, 11, 17, 4294967280UL) ^   // p3=2^28-1  
    LCGStep(z4, 1664525, 1013904223UL)        // p4=2^32  
   );
  //} while (!(randval > 0.0f && randval < 1.0f));
  return randval;
}  

__global__
static void _rand(float* vec, uint* z1, uint* z2, uint* z3, uint* z4) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  vec[i] = HybridTaus(z1[i],z2[i],z3[i],z4[i]);
}


//////////////////// xorshift1024*Phi Random Generator   /////////////

void initRand(uint64_t  seed)
    {
        xor_s[0] = seed;
        xor_s[1] = seed + seed;
        
        for ( int i = 0 ; i < 16 ; ++i )
        {
            xor_s[ i ] = nextRand();
        }
        xor_p = nextRand() % 15;
    }

uint64_t nextRand(void) {
	const uint64_t s0 = xor_s[xor_p];
	uint64_t s1 = xor_s[xor_p = (xor_p + 1) & 15];
	s1 ^= s1 << 31; // a
	xor_s[xor_p] = s1 ^ s0 ^ (s1 >> 11) ^ (s0 >> 30); // b,c
	return xor_s[xor_p] * 0x9e3779b97f4a7c13;
}

double uniformDoubleRand(void){
     return (double) nextRand() / (double) UINT64_MAX;
}


////
////  CORRELATOR SUBROUTINES
////

void initialize(int N, struct correlator mol[]) {
	
	int i,j,k,length=numcorrelators*p;
	
	for (k = 0; k < N; k ++) {
		for (j=0;j<numcorrelators;++j) {
			for (i=0;i<p;++i) {
				mol[k].shift[j][i] = -2E10;
				mol[k].correlation[j][i] = 0;
				mol[k].ncorrelation[j][i] = 0;
			}
			mol[k].accumulator[j] = 0.0;
			mol[k].naccumulator[j] = 0;
			mol[k].insertindex[j] = 0;
		}
	
		for (i=0;i<length;++i) {
			mol[k].t[i] = 0;
			mol[k].f[i] = 0;
		}
		
		mol[k].npcorr = 0;
		mol[k].kmax=0;
		mol[k].accval=0;
	}
}

void add(struct correlator mol[], int i, double w, int k) {
	int j, dmin = p/m_;
	
	/// If we exceed the correlator side, the value is discarded
	if (k == numcorrelators)
		return;
	if (k > mol[i].kmax) 
		mol[i].kmax = k;

	/// Insert new value in shift array
	mol[i].shift[k][mol[i].insertindex[k]] = w;

	/// Add to average value
	if (k==0)
		mol[i].accval += w;

	/// Add to accumulator and, if needed, add to next correlator
	mol[i].accumulator[k] += w;
	++mol[i].naccumulator[k];
	if (mol[i].naccumulator[k]==m_) {
		add(mol,i,mol[i].accumulator[k]/m_, k+1);
		mol[i].accumulator[k]=0;
		mol[i].naccumulator[k]=0;
	}

	/// Calculate correlation function
	int ind1 = mol[i].insertindex[k];
	
	if (k==0) { /// First correlator is different
		int ind2 = ind1;
		
		for (j=0;j<p;++j) {
			if (mol[i].shift[k][ind2] > -1e10) {
				mol[i].correlation[k][j] += mol[i].shift[k][ind1] * mol[i].shift[k][ind2];
				++mol[i].ncorrelation[k][j];
			}
			--ind2;
			if (ind2<0)
				ind2+=p;
		}
	}
	else {
		int ind2=ind1-dmin;
		for (j=dmin;j<p;++j) {
			if (ind2<0) 
				ind2+=p;
			if (mol[i].shift[k][ind2] > -1e10) {
				mol[i].correlation[k][j] += mol[i].shift[k][ind1] * mol[i].shift[k][ind2];
				++mol[i].ncorrelation[k][j];
			}
			--ind2;
		}
	}

	++ mol[i].insertindex[k];
	if (mol[i].insertindex[k]==p) 
		mol[i].insertindex[k]=0;
}

void evaluate(struct correlator mol[], int N, int norm) {
	int i,j,k,im,dmin=p/m_;

	double aux;
	
	for (j = 0; j < N; j ++) {
		
		aux = 0;
		im = 0;
		
		if (norm)
			aux = (mol[j].accval/mol[j].ncorrelation[0][0])*(mol[j].accval/mol[j].ncorrelation[0][0]);
	
		// First correlator
		for (i=0;i<p;++i) {
			if (mol[j].ncorrelation[0][i] > 0) {
				mol[j].t[im] = i;
				mol[j].f[im] = mol[j].correlation[0][i]/mol[j].ncorrelation[0][i] - aux;
				++im;
			}
		}
	
		// Subsequent correlators
		for (k=1;k<mol[j].kmax;++k) {
			for (i=dmin;i<p;++i) {
				if (mol[j].ncorrelation[k][i]>0) {
					mol[j].t[im] = i * pow((double)m_, k);
					mol[j].f[im] = mol[j].correlation[k][i] / mol[j].ncorrelation[k][i] - aux;
					++im;
				}
			}
		}
	
		mol[j].npcorr = im;
	
	}
}


void printCorr(struct correlator corr[], char filename[], int N) {
	
	FILE * fp;
//	char filename[100];
	int i,j;
	
//	sprintf(filename,"autocorr/autocorrelation-N%d-P%1.3f-%1.3f.dat",N,P,T);
	fp = fopen(filename,"w");
	
	double meanCorr[corr[0].npcorr];
	
	for (i = 0; i < corr[0].npcorr; i ++) {
		meanCorr[i] = 0;
		for(j = 0; j < N; j ++)
			meanCorr[i] += corr[j].f[i];
		meanCorr[i] /= (double) N;
		fprintf(fp,"%.0f\t%f\n",corr[0].t[i],meanCorr[i]/meanCorr[0]);
	}
	
	fclose(fp);
}


////
////  CHESSBOARD GPU ALGORITHM 
////


/****   EDGES AND VERTEX DEFINITION IN A CUBE

VERTEX DEFINITION

	Bottom layer (Z=0)       Y direction
                                  ^
    v3 [----------------] v8      [
       [                ]         [
       [                ]         [
       [                ]         [
       [                ]         [
       [                ]         [
    v1 [----------------] v2      [---------------> X direction

                                 v1 is placed at the origin of coordintates (X=0,Y=0,Z=0)
	Top layer (Z=1)

     v6	[----------------] v5
        [                ]
        [                ]
        [                ]
        [                ]
        [                ]
     v4 [----------------] v7

EDGES DEFINITION
	0	v1 -> v2
	1	v1 -> v3
	2	v1 -> v4
	3	v2 -> v7
	4	v2 -> v8
	5	v3 -> v6
	6	v3 -> v8
	7	v4 -> v6
	8	v4 -> v7
	9	v6 -> v5
	10	v7 -> v5
	11	v8 -> v5



****/


__global__ void gpu_set_cubes (int * devChessboard_edge, int * devChessboard_vertex)
{
    uint tid = blockIdx.x*blockDim.x + threadIdx.x;
	
    uint x = tid % NX;
    uint y = (tid/NX) % NY;
    uint z = tid/(NX*NY);


    if ( z%2 == 0 ){          // only survive threads with x, y, z all even or all odd
        if ( (x%2 != 0) || (y%2 != 0) ){
           return;
        }
    } else if ( z%2 == 1 ){
        if ( (x%2 != 1) || (y%2 != 1) ){
           return;
        }
    }

    int cube = (x/2) + (NX/2)*(y/2) + ((NX*NY)/4)*z;
    int NCUBES = (NX*NY*NZ/4);

//    if ( cube > NX*NY*NZ/4 )
//       return;

    for (int i=0; i < EDGES; i++)
             devChessboard_edge[cube+i*NCUBES] = 0;
    
    ////// set Vertex
     int v1, v2, v3, v4, v5, v6, v7, v8;

     v1 = x + y*NX + z*NX*NY;				// v1 = tid
     v2 = (x+1)%NX + y*NX + z*NX*NY;      		// v2 = xup ( v1 )
     v3 = x + ((y+1)%NY)*NX + z*NX*NY;    		// v3 = yup ( v1 )
     v4 = x + y*NX + ((z+1)%NZ)*NX*NY;    		// v4 = zup ( v1 )
     v6 = x + ((y+1)%NY)*NX + ((z+1)%NZ)*NX*NY;    	// v6 = zup ( v3 ) = zup ( yup ( v1 ) )
     v7 = (x+1)%NX + y*NX + ((z+1)%NZ)*NX*NY;		// v7 = zup ( v2 ) = zup ( xup ( v1 ) )
     v8 = (x+1)%NX + ((y+1)%NY)*NX + z*NX*NY;		// v8 = yup ( v2 ) = yup ( xup ( v1 ) )
     v5 = (x+1)%NX + ((y+1)%NY)*NX + ((z+1)%NZ)*NX*NY;  // v5 = xup ( v6 ) = xup ( zup ( yup ( v1 ) ) )
 
     devChessboard_vertex[cube+0*NCUBES+0*NCUBES*EDGES] = v1;
     devChessboard_vertex[cube+0*NCUBES+1*NCUBES*EDGES] = v2;

     devChessboard_vertex[cube+1*NCUBES+0*NCUBES*EDGES] = v1;
     devChessboard_vertex[cube+1*NCUBES+1*NCUBES*EDGES] = v3;

     devChessboard_vertex[cube+2*NCUBES+0*NCUBES*EDGES] = v1;
     devChessboard_vertex[cube+2*NCUBES+1*NCUBES*EDGES] = v4;

     devChessboard_vertex[cube+3*NCUBES+0*NCUBES*EDGES] = v2;
     devChessboard_vertex[cube+3*NCUBES+1*NCUBES*EDGES] = v7;

     devChessboard_vertex[cube+4*NCUBES+0*NCUBES*EDGES] = v2;
     devChessboard_vertex[cube+4*NCUBES+1*NCUBES*EDGES] = v8;

     devChessboard_vertex[cube+5*NCUBES+0*NCUBES*EDGES] = v3;
     devChessboard_vertex[cube+5*NCUBES+1*NCUBES*EDGES] = v6;

     devChessboard_vertex[cube+6*NCUBES+0*NCUBES*EDGES] = v3;
     devChessboard_vertex[cube+6*NCUBES+1*NCUBES*EDGES] = v8;

     devChessboard_vertex[cube+7*NCUBES+0*NCUBES*EDGES] = v4;
     devChessboard_vertex[cube+7*NCUBES+1*NCUBES*EDGES] = v6;

     devChessboard_vertex[cube+8*NCUBES+0*NCUBES*EDGES] = v4;
     devChessboard_vertex[cube+8*NCUBES+1*NCUBES*EDGES] = v7;

     devChessboard_vertex[cube+9*NCUBES+0*NCUBES*EDGES] = v6;
     devChessboard_vertex[cube+9*NCUBES+1*NCUBES*EDGES] = v5;

     devChessboard_vertex[cube+10*NCUBES+0*NCUBES*EDGES] = v7;
     devChessboard_vertex[cube+10*NCUBES+1*NCUBES*EDGES] = v5;

     devChessboard_vertex[cube+11*NCUBES+0*NCUBES*EDGES] = v8;
     devChessboard_vertex[cube+11*NCUBES+1*NCUBES*EDGES] = v5;
    //////

}

__global__ void gpu_Chessboard_start_cubes( int N, float * dev_Rnd_cubeFlip, int * devSolutions, byte * dev_active_bond,
  int * devChessboard_edge, int * devChessboard_vertex, int * devChessboard_state )   
// output ret in typewriter fashion
{

    uint tid = blockIdx.x*blockDim.x + threadIdx.x;
	
    uint x = tid % NX;
    uint y = (tid/NX) % NY;
    uint z = tid/(NX*NY);
 

    if ( z%2 == 0 ){          // only survive threads with x, y, z all even or all odd
        if ( (x%2 != 0) || (y%2 != 0) ){
           return;
        }
    }else if ( z%2 == 1 ){
        if ( (x%2 != 1) || (y%2 != 1) ){
           return;
        }
    }

   int cube = (x/2) + (NX/2)*(y/2) + ((NX*NY)/4)*z;
   int NCUBES = (NX*NY*NZ/4);

   int r = (int) (dev_Rnd_cubeFlip[cube]*NSOLUTIONS);
   r %= NSOLUTIONS;

   devChessboard_state[cube] = r;

    for(int i=0; i<EDGES; i++)
        devChessboard_edge[cube+i*NCUBES] = devSolutions[r*EDGES+i];
    
// Translate from chessboard to water bonding vector

    for (int j=0; j<EDGES; j ++){

        int v0 = devChessboard_vertex[cube+j*NCUBES+0*NCUBES*EDGES];
        int v1 = devChessboard_vertex[cube+j*NCUBES+1*NCUBES*EDGES];

        if ( j == 0 || j == 6 || j == 8 || j == 9 ){   // xup edges
              dev_active_bond[0*N+v0] = devChessboard_edge[cube+j*NCUBES];
              dev_active_bond[1*N+v1] = devChessboard_edge[cube+j*NCUBES];
        } else if ( j == 1 || j == 4 || j == 7 || j == 10 ){    // yup edges
              dev_active_bond[2*N+v0] = devChessboard_edge[cube+j*NCUBES];
              dev_active_bond[3*N+v1] = devChessboard_edge[cube+j*NCUBES];
        } else if ( j == 2 || j == 3 || j == 5 || j == 11 ){   // zup edges
              dev_active_bond[4*N+v0] = devChessboard_edge[cube+j*NCUBES];
              dev_active_bond[5*N+v1] = devChessboard_edge[cube+j*NCUBES];
        } 

    }

}

__global__ void gpu_Chessboard_set_state( int N, byte * dev_active_bond,
                int * devChessboard_vertex, int * devChessboard_state )   
// output ret in typewriter fashion
// sets cube state according to active bonds
{

    uint tid = blockIdx.x*blockDim.x + threadIdx.x;
	
    uint x = tid % NX;
    uint y = (tid/NX) % NY;
    uint z = tid/(NX*NY);
 

    if ( z%2 == 0 ){          // only survive threads with x, y, z all even or all odd
        if ( (x%2 != 0) || (y%2 != 0) ){
           return;
        }
    }else if ( z%2 == 1 ){
        if ( (x%2 != 1) || (y%2 != 1) ){
           return;
        }
    }

   int cube = (x/2) + (NX/2)*(y/2) + ((NX*NY)/4)*z;
   int NCUBES = (NX*NY*NZ/4);

   //cube vertex (cell id)
   int v1 = devChessboard_vertex[cube+0*NCUBES+0*NCUBES*EDGES];
   int v2 = devChessboard_vertex[cube+3*NCUBES+0*NCUBES*EDGES];
   int v3 = devChessboard_vertex[cube+5*NCUBES+0*NCUBES*EDGES];
   int v4 = devChessboard_vertex[cube+7*NCUBES+0*NCUBES*EDGES];
   int v5 = devChessboard_vertex[cube+7*NCUBES+1*NCUBES*EDGES];
   int v6 = devChessboard_vertex[cube+5*NCUBES+1*NCUBES*EDGES];
   int v7 = devChessboard_vertex[cube+3*NCUBES+1*NCUBES*EDGES];
   int v8 = devChessboard_vertex[cube+4*NCUBES+1*NCUBES*EDGES];

   if ( dev_active_bond[0*N+v1] == 0 ){

      if ( dev_active_bond[0*N+v6] == 0 ) devChessboard_state[cube] = 2;
      else if ( dev_active_bond[4*N+v3] == 0 ) devChessboard_state[cube] = 3;
      else devChessboard_state[cube] = 8;

   } else if ( dev_active_bond[2*N+v1] == 0 ){

      if ( dev_active_bond[4*N+v2] == 0 ) devChessboard_state[cube] = 4;
      else if ( dev_active_bond[2*N+v7] == 0 ) devChessboard_state[cube] = 5;
      else devChessboard_state[cube] = 7;
 
  } else {

      if ( dev_active_bond[0*N+v3] == 0 ) devChessboard_state[cube] = 0;
      else if ( dev_active_bond[4*N+v8] == 0 ) devChessboard_state[cube] = 1;
      else devChessboard_state[cube] = 6;

  }


}


__global__ void gpu_Chessboard_set_active_bonds(
   byte * dev_s, 
   int N, 
   double Jeff, 
   double beta, 
   float * dev_Rnd_cubeFlip, 
   float * dev_Rnd_chess,
   int offset_chess, 
   int * devSolutions,
   byte * dev_active_bond,
   int * devChessboard_edge,
   int * devChessboard_vertex,
   int * devChessboard_state )
{

    uint tid = blockIdx.x*blockDim.x + threadIdx.x;
	
    uint x = tid % NX;
    uint y = (tid/NX) % NY;
    uint z = tid/(NX*NY);

    if ( z%2 == 0 ){          // only survive threads with x, y, z all even or all odd
        if ( (x%2 != 0) || (y%2 != 0) ){
           return;
        }
    }else if ( z%2 == 1 ){
        if ( (x%2 != 1) || (y%2 != 1) ){
           return;
        }
    }

    int cube = (x/2) + (NX/2)*(y/2) + ((NX*NY)/4)*z;
    int NCUBES = NX*NY*NZ/4;

    int old_state = devChessboard_state[cube];

//    int new_state =  ( old_state + 1 + (int) ((NSOLUTIONS-1)*dev_Rnd_cubeFlip[cube+offset_chess]) ) % NSOLUTIONS; 
    int new_state = (int) (dev_Rnd_cubeFlip[cube+offset_chess]*NSOLUTIONS);
    new_state %= NSOLUTIONS;

    double delta_H=0;
			// Chessboard Metropolis
    for (int j=0; j<EDGES; j++){

        int node0 = devChessboard_vertex[cube+j*NCUBES+0*EDGES*NCUBES];
        int node1 = devChessboard_vertex[cube+j*NCUBES+1*EDGES*NCUBES];

        int arm0, arm1;
        if ( j == 0 || j == 6 || j == 8 || j == 9 ){   // xup edges
              arm0 = 0;
              arm1 = 1;
        } else if ( j == 1 || j == 4 || j == 7 || j == 10 ){    // yup edges
              arm0 = 2;
              arm1 = 3;
        } else if ( j == 2 || j == 3 || j == 5 || j == 11 ){   // zup edges
	      arm0 = 4;
              arm1 = 5;
        } 

        delta_H += ( devSolutions[new_state*EDGES+j] - devSolutions[old_state*EDGES+j] ) * delta(dev_s[arm0*N+node0],dev_s[arm1*N+node1]);
    }
    
    delta_H *= Jeff;

    if ( dev_Rnd_chess[cube+offset_chess] > expf(beta*delta_H) ){
              new_state = old_state;
    }
    
    devChessboard_state[cube] = new_state;
    for(int j=0; j<EDGES; j++)
        devChessboard_edge[cube+j*NCUBES] = devSolutions[new_state*EDGES+j];
    
// Translate from chessboard to water bonding vector

    for (int j=0; j<EDGES; j ++){

        int v0 = devChessboard_vertex[cube+j*NCUBES+0*EDGES*NCUBES];
        int v1 = devChessboard_vertex[cube+j*NCUBES+1*EDGES*NCUBES];

        if ( j == 0 || j == 6 || j == 8 || j == 9 ){   // xup edges
              dev_active_bond[0*N+v0] = devChessboard_edge[cube+j*NCUBES];
              dev_active_bond[1*N+v1] = devChessboard_edge[cube+j*NCUBES];
        } else if ( j == 1 || j == 4 || j == 7 || j == 10 ){    // yup edges
              dev_active_bond[2*N+v0] = devChessboard_edge[cube+j*NCUBES];
              dev_active_bond[3*N+v1] = devChessboard_edge[cube+j*NCUBES];
        } else if ( j == 2 || j == 3 || j == 5 || j == 11 ){   // zup edges
              dev_active_bond[4*N+v0] = devChessboard_edge[cube+j*NCUBES];
              dev_active_bond[5*N+v1] = devChessboard_edge[cube+j*NCUBES];
        } 

    }

}
